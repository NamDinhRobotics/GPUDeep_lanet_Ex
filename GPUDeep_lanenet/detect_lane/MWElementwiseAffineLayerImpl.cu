#include "MWElementwiseAffineLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "MWKernelHeaders.hpp"
#include "cnn_api.hpp"
#include <math.h>
#include <cassert>
#include <stdio.h>

MWElementwiseAffineLayerImpl::MWElementwiseAffineLayerImpl(MWCNNLayer *layer,
                                                           MWTargetNetworkImpl *ntwk_impl, int scale_H, int scale_W,
                                                           int scale_C, int
                                                           offset_H, int offset_W, int offset_C, bool isClipped,
                                                           int lowerbound, int
                                                           upperbound, const char *rISNTTiSXOTdHqHTtNiB, const char *
iADjqLChtuDbEWfMYFLp) : MWCNNLayerImpl(layer, ntwk_impl),
                        pvpNsgGssdTxeVoFIkXI(NULL), gNROjwaqhxDPvBWUCUcQ(NULL), qBTcAwVGZERyCjGYByPe(scale_H),
                        qWwjVYwfnvEnFKlgpqwA(scale_W), pzUAoBDvaKAtdsmkQuct(scale_C),
                        hljcfGWsvZXJZNrImpJB(offset_H), hvqKUzPqCuUJRfoNlbwW(offset_W),
                        hKyfKjPACkOBDvLdESxH(offset_C), ZUTPCvgISoRdtnhGqXzM(isClipped),
                        bQjijJlpNAVdwDDQgpaX(lowerbound), veFyKKHbdqBIvQLYBqfF(upperbound) {
    CUDA_CALL(hipMalloc((void **) &pvpNsgGssdTxeVoFIkXI,
                         sizeof(float) * qBTcAwVGZERyCjGYByPe * qWwjVYwfnvEnFKlgpqwA * pzUAoBDvaKAtdsmkQuct));
    CUDA_CALL(hipMalloc((void **) &gNROjwaqhxDPvBWUCUcQ,
                         sizeof(float) * hljcfGWsvZXJZNrImpJB * hvqKUzPqCuUJRfoNlbwW * hKyfKjPACkOBDvLdESxH));
    loadScale(rISNTTiSXOTdHqHTtNiB);
    loadOffset(iADjqLChtuDbEWfMYFLp);
}

MWElementwiseAffineLayerImpl::~MWElementwiseAffineLayerImpl() {}

void
MWElementwiseAffineLayerImpl::propagateSize() {}

void
MWElementwiseAffineLayerImpl::predict() {
    MWTensorBase *ipTensorBase =
            getLayer()->getInputTensor(0);
    MWTensorBase *opTensorBase =
            getLayer()->getOutputTensor(0);
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    int WmXADZOqdcQvtBUvFerh =
            ipTensor->getHeight();
    int WprSrhAStKGxyXeoxETy = ipTensor->getWidth();
    int
            WerBmCOBWhvoFbdqfitc = ipTensor->getChannels();
    long int
            YOWMnLKOMqAODXiVNoGy = WmXADZOqdcQvtBUvFerh * WprSrhAStKGxyXeoxETy;
    long
    int YNmJhGSUszJKxsodxiuV =
            YOWMnLKOMqAODXiVNoGy * WerBmCOBWhvoFbdqfitc;
    long int
            YNDVziqpDddiXQKYZZhX = ipTensor->getNumElements();
    long int sFIUeCwGDlfadqOrGZHC =
            ((YNDVziqpDddiXQKYZZhX + 31) / 32) * 32;
    int tGsvtyAVkrDznETdweDC =
            (sFIUeCwGDlfadqOrGZHC < 1024) ? sFIUeCwGDlfadqOrGZHC : 1024;
    long int
            KHClOltUSuqFVVErSxVb = (YNDVziqpDddiXQKYZZhX + tGsvtyAVkrDznETdweDC -
                                    1) / tGsvtyAVkrDznETdweDC;
    long int qEXwbWWsnOADJeTXfRVa =
            qBTcAwVGZERyCjGYByPe * qWwjVYwfnvEnFKlgpqwA * pzUAoBDvaKAtdsmkQuct;
    long int
            hnewnpwgzKmOdualajhn = hljcfGWsvZXJZNrImpJB * hvqKUzPqCuUJRfoNlbwW *
                                   hKyfKjPACkOBDvLdESxH;
    assert(qEXwbWWsnOADJeTXfRVa <= YNDVziqpDddiXQKYZZhX);
    assert(hnewnpwgzKmOdualajhn <= YNDVziqpDddiXQKYZZhX);
    if (qEXwbWWsnOADJeTXfRVa ==
        1) {
        scale_scalar_kernel<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(ipTensor->getData(), opTensor->getData(),
                                pvpNsgGssdTxeVoFIkXI, YNDVziqpDddiXQKYZZhX);
    } else if (qBTcAwVGZERyCjGYByPe == 1 &&
               qWwjVYwfnvEnFKlgpqwA == 1 && qEXwbWWsnOADJeTXfRVa > 1) {
        scale_vector_kernel<<<KHClOltUSuqFVVErSxVb, tGsvtyAVkrDznETdweDC>>>(
                ipTensor->getData(), opTensor->getData(), pvpNsgGssdTxeVoFIkXI,
                YOWMnLKOMqAODXiVNoGy, YNmJhGSUszJKxsodxiuV,
                YNDVziqpDddiXQKYZZhX);
    } else if (YNmJhGSUszJKxsodxiuV ==
               qEXwbWWsnOADJeTXfRVa) {
        scale_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(ipTensor->getData(), opTensor->getData(),
                                pvpNsgGssdTxeVoFIkXI, YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX);
    } else {
        scale_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(ipTensor->getData(), opTensor->getData(),
                                pvpNsgGssdTxeVoFIkXI, YOWMnLKOMqAODXiVNoGy, YNDVziqpDddiXQKYZZhX);
    }
    if
            (hnewnpwgzKmOdualajhn == 1) {
        offset_scalar_kernel<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(opTensor->getData(), opTensor->getData(),
                                gNROjwaqhxDPvBWUCUcQ, YNDVziqpDddiXQKYZZhX, ZUTPCvgISoRdtnhGqXzM,
                                bQjijJlpNAVdwDDQgpaX, veFyKKHbdqBIvQLYBqfF);
    } else if (hljcfGWsvZXJZNrImpJB
               == 1 && hvqKUzPqCuUJRfoNlbwW == 1 && hnewnpwgzKmOdualajhn > 1) {
        offset_vector_kernel<<<KHClOltUSuqFVVErSxVb, tGsvtyAVkrDznETdweDC>>>(
                opTensor->getData(), opTensor->getData(), gNROjwaqhxDPvBWUCUcQ,
                YOWMnLKOMqAODXiVNoGy, YNmJhGSUszJKxsodxiuV,
                YNDVziqpDddiXQKYZZhX, ZUTPCvgISoRdtnhGqXzM, bQjijJlpNAVdwDDQgpaX,
                veFyKKHbdqBIvQLYBqfF);
    } else if (YNmJhGSUszJKxsodxiuV ==
               hnewnpwgzKmOdualajhn) {
        offset_tensor3d_kernel<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(opTensor->getData(), opTensor->getData(),
                                gNROjwaqhxDPvBWUCUcQ, YNmJhGSUszJKxsodxiuV, YNDVziqpDddiXQKYZZhX,
                                ZUTPCvgISoRdtnhGqXzM, bQjijJlpNAVdwDDQgpaX, veFyKKHbdqBIvQLYBqfF);
    } else {
        offset_matrix2d_kernel<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(opTensor->getData(), opTensor->getData(),
                                gNROjwaqhxDPvBWUCUcQ, YOWMnLKOMqAODXiVNoGy, YNDVziqpDddiXQKYZZhX,
                                ZUTPCvgISoRdtnhGqXzM, bQjijJlpNAVdwDDQgpaX, veFyKKHbdqBIvQLYBqfF);
    }
    return;
}

void MWElementwiseAffineLayerImpl::cleanup() {
    if (pvpNsgGssdTxeVoFIkXI) {
        CUDA_FREE_CALL(pvpNsgGssdTxeVoFIkXI);
        pvpNsgGssdTxeVoFIkXI = NULL;
    }
    if
            (gNROjwaqhxDPvBWUCUcQ) {
        CUDA_FREE_CALL(gNROjwaqhxDPvBWUCUcQ);
        gNROjwaqhxDPvBWUCUcQ =
                NULL;
    }
}

void MWElementwiseAffineLayerImpl::loadScale(const char *
rISNTTiSXOTdHqHTtNiB) {
    FILE *QjgQHaUACFNSteMrRtRj =
            MWCNNLayer::openBinaryFile(rISNTTiSXOTdHqHTtNiB);
    assert(QjgQHaUACFNSteMrRtRj);
    long
    int dkLDkRwCBjeybwDHbKiE = qBTcAwVGZERyCjGYByPe * qWwjVYwfnvEnFKlgpqwA * pzUAoBDvaKAtdsmkQuct;
    float *KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * dkLDkRwCBjeybwDHbKiE);
    call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dkLDkRwCBjeybwDHbKiE, QjgQHaUACFNSteMrRtRj,
               rISNTTiSXOTdHqHTtNiB);
    CUDA_CALL(hipMemcpy(pvpNsgGssdTxeVoFIkXI,
                         KZWeXiYFmdpQdsgidKeG, sizeof(float) * dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice));
    free(KZWeXiYFmdpQdsgidKeG);
    fclose(QjgQHaUACFNSteMrRtRj);
}

void
MWElementwiseAffineLayerImpl::loadOffset(const char *iADjqLChtuDbEWfMYFLp) {
    FILE *QjgQHaUACFNSteMrRtRj = MWCNNLayer::openBinaryFile(iADjqLChtuDbEWfMYFLp);
    assert(QjgQHaUACFNSteMrRtRj);
    long int dkLDkRwCBjeybwDHbKiE =
            hljcfGWsvZXJZNrImpJB * hvqKUzPqCuUJRfoNlbwW * hKyfKjPACkOBDvLdESxH;
    float *
            KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * dkLDkRwCBjeybwDHbKiE);
    call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dkLDkRwCBjeybwDHbKiE, QjgQHaUACFNSteMrRtRj,
               iADjqLChtuDbEWfMYFLp);
    CUDA_CALL(hipMemcpy(gNROjwaqhxDPvBWUCUcQ,
                         KZWeXiYFmdpQdsgidKeG, sizeof(float) * dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice));
    free(KZWeXiYFmdpQdsgidKeG);
    fclose(QjgQHaUACFNSteMrRtRj);
}