#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cassert>
#include <stdio.h>
#include <stdexcept>
#include "MWCNNLayerImpl.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"

#ifdef RANDOM
#include <hiprand.h>
hiprandGenerator_t REXdEoRjxuQJkqgIDihy; void
curand_call_line_file(hiprandStatus_t rlQsibXJSWJVnUVpdNeL, const int 
bDTIjtxZiSHtjwzgEluE, const char *QMgBqCuvjnbWHWiVPEwn) { if (rlQsibXJSWJVnUVpdNeL != 
HIPRAND_STATUS_SUCCESS) { char buffer[100]; int numElem = sprintf(buffer, 
"%d at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL, bDTIjtxZiSHtjwzgEluE, 
QMgBqCuvjnbWHWiVPEwn); throw std::runtime_error(buffer); } }
#endif

float *malloc_call_line_file(size_t msize, const int bDTIjtxZiSHtjwzgEluE, const
char *QMgBqCuvjnbWHWiVPEwn) {
    float *mem = (float *) malloc(msize);
    if (!mem) {
        char
                buffer[100];
        int numElem = sprintf(buffer, "%s at line: %d, file: %s\n",
                              "Memory allocation failed. ", bDTIjtxZiSHtjwzgEluE, QMgBqCuvjnbWHWiVPEwn);
        throw
                std::runtime_error(buffer);
    }
    return mem;
}

void
cuda_call_line_file(hipError_t rlQsibXJSWJVnUVpdNeL, const int bDTIjtxZiSHtjwzgEluE,
                    const char *QMgBqCuvjnbWHWiVPEwn) {
    if (rlQsibXJSWJVnUVpdNeL != hipSuccess) {
        throw_cuda_error(rlQsibXJSWJVnUVpdNeL, bDTIjtxZiSHtjwzgEluE, QMgBqCuvjnbWHWiVPEwn);
    }
}

void throw_cuda_error(hipError_t rlQsibXJSWJVnUVpdNeL, const int bDTIjtxZiSHtjwzgEluE,
                      const char *QMgBqCuvjnbWHWiVPEwn) {
    char buffer[100];
    int numElem = sprintf(buffer,
                          "Cuda Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL,
                          hipGetErrorString(rlQsibXJSWJVnUVpdNeL), bDTIjtxZiSHtjwzgEluE, QMgBqCuvjnbWHWiVPEwn);
    rlQsibXJSWJVnUVpdNeL = hipGetLastError();
    throw std::runtime_error(buffer);
}

void cudnn_call_line_file(hipdnnStatus_t rlQsibXJSWJVnUVpdNeL, const int
bDTIjtxZiSHtjwzgEluE, const char *QMgBqCuvjnbWHWiVPEwn) {
    if (rlQsibXJSWJVnUVpdNeL !=
        HIPDNN_STATUS_SUCCESS) {
        char buffer[100];
        int numElem = sprintf(buffer,
                              "CuDNN Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL,
                              hipdnnGetErrorString(rlQsibXJSWJVnUVpdNeL), bDTIjtxZiSHtjwzgEluE, QMgBqCuvjnbWHWiVPEwn);
        throw std::runtime_error(buffer);
    }
}

const char *
cublasGetErrorString(hipblasStatus_t rlQsibXJSWJVnUVpdNeL) {
    switch (rlQsibXJSWJVnUVpdNeL) {
        case HIPBLAS_STATUS_SUCCESS:
            return
                    "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return
                    "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED:
            return
                    "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE:
            return
                    "HIPBLAS_STATUS_INVALID_VALUE";
        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return
                    "HIPBLAS_STATUS_ARCH_MISMATCH";
        case HIPBLAS_STATUS_MAPPING_ERROR:
            return
                    "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return
                    "HIPBLAS_STATUS_EXECUTION_FAILED";
        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return
                    "HIPBLAS_STATUS_INTERNAL_ERROR";
        case HIPBLAS_STATUS_NOT_SUPPORTED:
            return
                    "HIPBLAS_STATUS_NOT_SUPPORTED";
        case HIPBLAS_STATUS_UNKNOWN:
            return
                    "HIPBLAS_STATUS_UNKNOWN";
    }
    return "unknown error";
}

void
cublas_call_line_file(hipblasStatus_t rlQsibXJSWJVnUVpdNeL, const int
bDTIjtxZiSHtjwzgEluE, const char *QMgBqCuvjnbWHWiVPEwn) {
    if (rlQsibXJSWJVnUVpdNeL !=
        HIPBLAS_STATUS_SUCCESS) {
        char buffer[100];
        int numElem = sprintf(buffer,
                              "CuBlas Error %d(%s) at line: %d, file: %s\n", rlQsibXJSWJVnUVpdNeL,
                              cublasGetErrorString(rlQsibXJSWJVnUVpdNeL), bDTIjtxZiSHtjwzgEluE, QMgBqCuvjnbWHWiVPEwn);
        throw std::runtime_error(buffer);
    }
}

MWCNNLayerImpl::MWCNNLayerImpl(MWCNNLayer *layer, MWTargetNetworkImpl *
ntwk_impl) : PfisSEEWDaQFynnzlcin(0.0), OzygUJRIZYnGLzSjgahB(1.0),
             OwenhowBxTAXHXmJpIKd(-1.0), atVCyzqXZAZxwlkRLBRA(layer),
             dMxIKDGTITyhdLqIHBLA(ntwk_impl) {}

MWCNNLayerImpl::~MWCNNLayerImpl() {
    for (std::map<int, hipdnnTensorDescriptor_t *>::iterator it =
            lHtftnmGBvlSSoGOXVui.begin(); it != lHtftnmGBvlSSoGOXVui.end(); ++it) {
        CUDNN_CALL(hipdnnDestroyTensorDescriptor(*it->second));
        delete it->second;
        it->second = 0;
    }
}

template<class T>
void
MWCNNLayerImpl::allocateOutputData(int outIdx) {
    MWTensorBase *opTensorBase =
            getLayer()->getOutputTensor(outIdx);
    bool bufferReuse =
            opTensorBase->getopBufIndex() >= 0;
    if (bufferReuse) {
        assert(opTensorBase->isFloat());
        MWTensor<float> *opTensor =
                static_cast<MWTensor<float> *>(opTensorBase);
        opTensor->setData(dMxIKDGTITyhdLqIHBLA->memBuffer[opTensor->getopBufIndex()]);
    } else {
        int inIdx = getLayer()->getInPlaceIndex(outIdx);
        if (inIdx != -1) {
            MWTensor<T> *ipTensor =
                    static_cast<MWTensor<T> *>(getLayer()->getInputTensor(inIdx));
            MWTensor<T> *
                    opTensor = static_cast<MWTensor<T> *>(opTensorBase);
            T *ipData =
                    ipTensor->getData();
            assert(ipData);
            opTensor->setData(ipData);
        } else {
            MWTensor<T> *opTensor = static_cast<MWTensor<T> *>(opTensorBase);
            T *
                    OKaRVOctKLlnIyGmjRNW;
            CUDA_CALL(hipMalloc((void **) &OKaRVOctKLlnIyGmjRNW,
                                 sizeof(T) * opTensor->getNumElements()));
            opTensor->setData(OKaRVOctKLlnIyGmjRNW);
        }
    }
}

template void MWCNNLayerImpl::allocateOutputData<float>(int);

template void
MWCNNLayerImpl::allocateOutputData<signed char>(int);

template<class T>
void
MWCNNLayerImpl::deallocateOutputData(int outIdx) {
    if (getLayer()->getInPlaceIndex(outIdx) == -1) {
        MWTensor<T> *opTensor =
                static_cast<MWTensor<T> *>(getLayer()->getOutputTensor(outIdx));
        T *data =
                opTensor->getData();
        CUDA_FREE_CALL(data);
    }
}

template void
MWCNNLayerImpl::deallocateOutputData<float>(int);

template void
MWCNNLayerImpl::deallocateOutputData<signed char>(int);

float *
MWCNNLayerImpl::getZeroPtr() { return &PfisSEEWDaQFynnzlcin; }

float *
MWCNNLayerImpl::getOnePtr() { return &OzygUJRIZYnGLzSjgahB; }

float *
MWCNNLayerImpl::getNegOnePtr() { return &OwenhowBxTAXHXmJpIKd; }

hipdnnTensorDescriptor_t *MWCNNLayerImpl::createAndAddDescriptor(int index) {
    std::map<int, hipdnnTensorDescriptor_t *>::iterator it =
            lHtftnmGBvlSSoGOXVui.find(index);
    assert(it == lHtftnmGBvlSSoGOXVui.end());
    hipdnnTensorDescriptor_t *newDescriptor = new hipdnnTensorDescriptor_t;
    if
            (!newDescriptor) { MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__); }
    lHtftnmGBvlSSoGOXVui[index] = newDescriptor;
    CUDNN_CALL(hipdnnCreateTensorDescriptor(newDescriptor));
    return newDescriptor;
}

hipdnnTensorDescriptor_t *MWCNNLayerImpl::getDescriptor(int index) {
    std::map<int, hipdnnTensorDescriptor_t *>::iterator it =
            lHtftnmGBvlSSoGOXVui.find(index);
    if (it != lHtftnmGBvlSSoGOXVui.end()) {
        return it->second;
    } else { return NULL; }
}

template<class T>
void
MWCNNLayerImpl::setDescriptor(hipdnnTensorDescriptor_t &desc, MWTensor<T> *
tensor) {
    if (tensor->getSequenceLength() == 1) {
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW,
                                              MWCNNLayerImpl::getCuDNNDataType<T>(), tensor->getBatchSize(),
                                              tensor->getChannels(), tensor->getHeight(), tensor->getWidth()));
    } else {
        int dims[5] = {tensor->getSequenceLength(), tensor->getBatchSize(),
                       tensor->getChannels(), tensor->getHeight(), tensor->getWidth()};
        int
                strides[5];
        MWTensorBase::getStrides(dims, 5, strides);
        CUDNN_CALL(hipdnnSetTensorNdDescriptor(desc,
                                              MWCNNLayerImpl::getCuDNNDataType<T>(), 5, dims, strides));
    }
}

template void
MWCNNLayerImpl::setDescriptor<float>(hipdnnTensorDescriptor_t &,
                                     MWTensor<float> *);

template void MWCNNLayerImpl::setDescriptor<signed
char>(hipdnnTensorDescriptor_t &, MWTensor<signed char> *);

template<>
hipdnnDataType_t MWCNNLayerImpl::getCuDNNDataType<float>() {
    return
            HIPDNN_DATA_FLOAT;
}

template<>
hipdnnDataType_t
MWCNNLayerImpl::getCuDNNDataType<signed char>() { return HIPDNN_DATA_INT8; }

hipdnnTensorDescriptor_t MWCNNLayerImpl::getCuDNNDescriptor(MWTensorBase *
tensor) {
    MWCNNLayer *layer = tensor->getOwner();
    MWCNNLayerImpl *impl =
            layer->getImpl();
    if (impl) {
        hipdnnTensorDescriptor_t *desc =
                impl->getDescriptor(tensor->getSourcePortIndex());
        if (desc == NULL) {
            impl->createAndAddDescriptor(tensor->getSourcePortIndex());
            desc =
                    impl->getDescriptor(tensor->getSourcePortIndex());
            assert(desc);
        }
        if
                (tensor->isFloat()) {
            MWCNNLayerImpl::setDescriptor<float>(*desc,
                                                 static_cast<MWTensor<float> *>(tensor));
        } else {
            assert(tensor->isInt8());
            MWCNNLayerImpl::setDescriptor<signed char>(*desc, static_cast<MWTensor<signed
            char> *>(tensor));
        }
        return *desc;
    } else {
        hipdnnTensorDescriptor_t
                tmpDescriptor;
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&tmpDescriptor));
        if
                (tensor->isFloat()) {
            MWCNNLayerImpl::setDescriptor<float>(tmpDescriptor,
                                                 static_cast<MWTensor<float> *>(tensor));
        } else {
            assert(tensor->isInt8());
            MWCNNLayerImpl::setDescriptor<signed char>(tmpDescriptor,
                                                       static_cast<MWTensor<signed char> *>(tensor));
        }
        return tmpDescriptor;
    }
}

void
__global__ __launch_bounds__(1024) padInputImpl(float *in, int inputH, int
inputW, int inputCh, int outputH, int outputW, int offsetH, int offsetW, float *
out, int inputElems) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i <
                                                        inputElems; i += blockDim.x * gridDim.x) {
        int idxB = i / (inputH * inputW * inputCh);
        int rem = (i - idxB * (inputH * inputW * inputCh));
        int idxCh = rem / (inputH * inputW);
        int rem1 = rem - idxCh * (inputH * inputW);
        int idxH = rem1 / inputW;
        int idxCol =
                rem1 - idxH * inputW;
        if ((idxH < inputH) && (idxCol < inputW)) {
            int outputR =
                    idxH + offsetH;
            int outputCol = idxCol + offsetW;
            int outputCh = inputCh;
            out[idxB * (outputH * outputW * outputCh) + idxCh * (outputH * outputW) +
                outputR * (outputW) + outputCol] = in[i];
        }
    }
}

void
MWCNNLayerImpl::padInput(float *TbrNrGxaFFHrzKUcfHNZ, int VenwEUlYwOBrwLVUhgUH, int
WOJynDmqVUPWjAGVIuMQ, int VFKMunbyHoAmpHUSkuUn, int lWJYwWaFPmWNQDPrlqER, int
                         lsqeARVLtpJTWezgnTkg, int gWETwFdWHfKuelmlKNCC, int hDaNSVZAofAENeIAiWEw, float *
jscBrjkVJyVfMMDjFpgl, int eqOmMKQRpqBqRQCnJmxt) {
    int tGsvtyAVkrDznETdweDC =
            (eqOmMKQRpqBqRQCnJmxt + 31) / 32 * 32;
    tGsvtyAVkrDznETdweDC =
            (tGsvtyAVkrDznETdweDC < 1024) ? tGsvtyAVkrDznETdweDC : 1024;
    int
            KHClOltUSuqFVVErSxVb = (eqOmMKQRpqBqRQCnJmxt + tGsvtyAVkrDznETdweDC -
                                    1) / tGsvtyAVkrDznETdweDC;
    padInputImpl<<<KHClOltUSuqFVVErSxVb,
    tGsvtyAVkrDznETdweDC>>>(TbrNrGxaFFHrzKUcfHNZ, VenwEUlYwOBrwLVUhgUH,
                            WOJynDmqVUPWjAGVIuMQ, VFKMunbyHoAmpHUSkuUn, lWJYwWaFPmWNQDPrlqER, lsqeARVLtpJTWezgnTkg,
                            gWETwFdWHfKuelmlKNCC, hDaNSVZAofAENeIAiWEw, jscBrjkVJyVfMMDjFpgl, eqOmMKQRpqBqRQCnJmxt);
}

void __global__ __launch_bounds__(1024) fillOutputBufferImpl(signed char *in,
                                                             int inputH, int inputW, int inputCh, int outputH,
                                                             int outputW, int offsetH, int
                                                             offsetW, signed char *out, int inputElems, int outputCh) {
    for (int i =
            blockDim.x * blockIdx.x + threadIdx.x; i < inputElems; i +=
                                                                           blockDim.x * gridDim.x) {
        int idxB = i / (inputH * inputW * inputCh);
        int rem = (i -
                   idxB * (inputH * inputW * inputCh));
        int idxCh = rem / (inputH * inputW);
        int rem1 = rem
                   - idxCh * (inputH * inputW);
        int idxH = rem1 / inputW;
        int idxCol = rem1 -
                     idxH * inputW;
        if ((idxH < inputH) && (idxCol < inputW)) {
            int outputR = idxH +
                          offsetH;
            int outputCol = idxCol + offsetW;
            *(out +
              idxB * (outputH * outputW * outputCh) + idxCh * (outputH * outputW) + outputR * (outputW) +
              outputCol) = *(in + i);
        }
    }
}

void MWCNNLayerImpl::fillOutputBuffer(signed
                                      char *TbrNrGxaFFHrzKUcfHNZ, int VenwEUlYwOBrwLVUhgUH, int WOJynDmqVUPWjAGVIuMQ,
                                      int
                                      VFKMunbyHoAmpHUSkuUn, int lWJYwWaFPmWNQDPrlqER, int lsqeARVLtpJTWezgnTkg, int
                                      gWETwFdWHfKuelmlKNCC, int hDaNSVZAofAENeIAiWEw, signed char *jscBrjkVJyVfMMDjFpgl,
                                      int
                                      eqOmMKQRpqBqRQCnJmxt, int kqftrrQBBOgGsrDSkIUk) {
    int tGsvtyAVkrDznETdweDC
            = (eqOmMKQRpqBqRQCnJmxt < 1024) ? eqOmMKQRpqBqRQCnJmxt : 1024;
    int
            KHClOltUSuqFVVErSxVb = (eqOmMKQRpqBqRQCnJmxt + tGsvtyAVkrDznETdweDC -
                                    1) / tGsvtyAVkrDznETdweDC;
    fillOutputBufferImpl<<<KHClOltUSuqFVVErSxVb,
    tGsvtyAVkrDznETdweDC>>>(TbrNrGxaFFHrzKUcfHNZ, VenwEUlYwOBrwLVUhgUH,
                            WOJynDmqVUPWjAGVIuMQ, VFKMunbyHoAmpHUSkuUn, lWJYwWaFPmWNQDPrlqER, lsqeARVLtpJTWezgnTkg,
                            gWETwFdWHfKuelmlKNCC, hDaNSVZAofAENeIAiWEw, jscBrjkVJyVfMMDjFpgl, eqOmMKQRpqBqRQCnJmxt,
                            kqftrrQBBOgGsrDSkIUk);
}

void MWCNNLayerImpl::throwAllocationError(const int
                                          line, const char *file) {
    char buffer[200];
    int numElem = sprintf(buffer,
                          "Failed to allocate memory at %d, file %s\n", line, file);
    throw
            std::runtime_error(buffer);
}

MWReLULayerImpl::MWReLULayerImpl(MWCNNLayer *
layer, MWTargetNetworkImpl *ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) {
    CUDNN_CALL(hipdnnCreateActivationDescriptor(&oKIvzXXMucEDsTGGpdpm));
    createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
}

MWReLULayerImpl::~MWReLULayerImpl() {}

void MWReLULayerImpl::propagateSize() {
    MWTensorBase *opTensor = getLayer()->getOutputTensor(0);
    hipdnnTensorDescriptor_t *desc = getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    setDescriptor<float>(*desc,
                         static_cast<MWTensor<float> *>(opTensor));
    CUDNN_CALL(hipdnnSetActivationDescriptor(oKIvzXXMucEDsTGGpdpm,
                                            HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));
}

void
MWReLULayerImpl::predict() {
    MWReLULayer *reluLayer =
            static_cast<MWReLULayer *>(getLayer());
    MWTensorBase *ipTensorBase =
            reluLayer->getInputTensor(0);
    MWTensorBase *opTensorBase =
            reluLayer->getOutputTensor(0);
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    hipdnnTensorDescriptor_t ipDesc =
            MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase);
    CUDNN_CALL(hipdnnActivationForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                      oKIvzXXMucEDsTGGpdpm, getOnePtr(), ipDesc, ipTensor->getData(), getZeroPtr(),
                                      *desc, opTensor->getData()));
}

void MWReLULayerImpl::cleanup() {
    CUDNN_CALL(hipdnnDestroyActivationDescriptor(oKIvzXXMucEDsTGGpdpm));
}

MWNormLayerImpl::MWNormLayerImpl(MWCNNLayer *layer, MWTargetNetworkImpl *
ntwk_impl, unsigned GFggoMvRWucDMqzlWzCl, double AHqhysOOIgbDpWZoPUFT,
                                 double AIXLuRgdeiqpaCehGSYD, double BdqURaHPmdnfzvtUvocl) : MWCNNLayerImpl(layer,
                                                                                                            ntwk_impl) {
    CUDNN_CALL(hipdnnCreateLRNDescriptor(&dJcdBfQQLhIAYHPxwQeg));
    createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
    CUDNN_CALL(hipdnnSetLRNDescriptor(dJcdBfQQLhIAYHPxwQeg,
                                     GFggoMvRWucDMqzlWzCl, AHqhysOOIgbDpWZoPUFT, AIXLuRgdeiqpaCehGSYD,
                                     BdqURaHPmdnfzvtUvocl));
}

MWNormLayerImpl::~MWNormLayerImpl() {}

void
MWNormLayerImpl::propagateSize() {
    MWTensorBase *opTensor =
            getLayer()->getOutputTensor(0);
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    setDescriptor<float>(*desc, static_cast<MWTensor<float> *>(opTensor));
}

void
MWNormLayerImpl::predict() {
    MWNormLayer *normLayer =
            static_cast<MWNormLayer *>(getLayer());
    MWTensorBase *ipTensorBase =
            normLayer->getInputTensor();
    MWTensorBase *opTensorBase =
            normLayer->getOutputTensor();
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    hipdnnTensorDescriptor_t ipDesc =
            MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase);
    CUDNN_CALL(hipdnnLRNCrossChannelForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                           dJcdBfQQLhIAYHPxwQeg, HIPDNN_LRN_CROSS_CHANNEL, getOnePtr(), ipDesc,
                                           ipTensor->getData(), getZeroPtr(), *desc, opTensor->getData()));
}

void
MWNormLayerImpl::cleanup() {
    CUDNN_CALL(hipdnnDestroyLRNDescriptor(dJcdBfQQLhIAYHPxwQeg));
}

void __global__
MWSetDyForBackPropImpl(float *PVBPDNaynqYkBlDZgXgj, const int fOpFYwKNwIfWjnPzNuob);

void __global__ doMWMaxPoolingLayerImpl(float *UdmcwaUkepxfZrpdpcAN,
                                        float *UWAGLbDcvybdWBtshhsr, const int BkwhtPQUCQKchmmimoXs);

MWMaxPoolingLayerImpl::MWMaxPoolingLayerImpl(MWCNNLayer *layer,
                                             MWTargetNetworkImpl *ntwk_impl, int DqxLTLaJwwgQqmrtCDuu, int
                                             EvebzoroiuKkIxwjkGnD, int FshVHIJMRAhtQirYPlZd, int
                                             GDRXdUDklKFEYEfifhIH, int CpMjJjtGOeWOzwxpAAQP, int
                                             ClEhcJFlvGCgiavziIag, int DCdZnqpcBnvXVgEsLBnz, int
                                             DGzdAcREJHGXjyRzNjJV, bool GIbahSoBBDrvvZduPEqU, int fSKMHAqIghbYYgyIpNDw)
        : MWCNNLayerImpl(layer, ntwk_impl),
          BUOdotSvmFyUWQKMUdra(GIbahSoBBDrvvZduPEqU), UdmcwaUkepxfZrpdpcAN(0), PVBPDNaynqYkBlDZgXgj(0),
          DSsxcjIrUgZCKZovyNQf(DqxLTLaJwwgQqmrtCDuu),
          EfvWctmlsWAPsxXgdKWf(EvebzoroiuKkIxwjkGnD),
          DRzwhbNPpftRRIXXfHzd(DqxLTLaJwwgQqmrtCDuu),
          ECTnqgWHyHCHCLBZlffd(EvebzoroiuKkIxwjkGnD),
          CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP),
          CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag),
          CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz),
          CufLFODQDXTAPyRqYodN(DGzdAcREJHGXjyRzNjJV),
          FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd),
          FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH),
          fXhhiexIRPLyKXApPmmy(fSKMHAqIghbYYgyIpNDw) {
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&muwRQxtWMMXAPxSuMYBw));
    createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
}

MWMaxPoolingLayerImpl::~MWMaxPoolingLayerImpl() {}

void
MWMaxPoolingLayerImpl::propagateSize() {
    MWTensorBase *ipTensor =
            getLayer()->getInputTensor(0);
    MWTensorBase *opTensor =
            getLayer()->getOutputTensor(0);
    if ((DSsxcjIrUgZCKZovyNQf == -1) &&
        (EfvWctmlsWAPsxXgdKWf == -1)) {
        DRzwhbNPpftRRIXXfHzd = ipTensor->getHeight();
        ECTnqgWHyHCHCLBZlffd = ipTensor->getWidth();
    }
    int nDsbARncmIrIaLubvLVZ =
            CZNYmBcNFSZWvaCklqeM;
    int nNULvWnBXnnWdpEkHPAH =
            CqtPRJvHlGJFssiPzsOm;
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(muwRQxtWMMXAPxSuMYBw, HIPDNN_POOLING_MAX,
                                           HIPDNN_NOT_PROPAGATE_NAN, DRzwhbNPpftRRIXXfHzd, ECTnqgWHyHCHCLBZlffd,
                                           nDsbARncmIrIaLubvLVZ, nNULvWnBXnnWdpEkHPAH, FrpxvsDMwwgbpqHXWxmN,
                                           FwLnexHgxHRquTKmNpoa));
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    setDescriptor<float>(*desc, static_cast<MWTensor<float> *>(opTensor));
}

void
MWMaxPoolingLayerImpl::allocate() {
    MWMaxPoolingLayer *maxpoolLayer =
            static_cast<MWMaxPoolingLayer *>(getLayer());
    MWTensorBase *ipTensor =
            maxpoolLayer->getInputTensor(0);
    MWTensorBase *opTensor =
            maxpoolLayer->getOutputTensor(0);
    if (BUOdotSvmFyUWQKMUdra) {
        const int
                edQOkUJIZbwzEeIcCLzG = ipTensor->getNumElements();
        CUDA_CALL(hipMalloc((void **) &UdmcwaUkepxfZrpdpcAN,
                             sizeof(float) * edQOkUJIZbwzEeIcCLzG));
        const int fOpFYwKNwIfWjnPzNuob =
                opTensor->getNumElements();
        CUDA_CALL(hipMalloc((void **) &PVBPDNaynqYkBlDZgXgj,
                             sizeof(float) * fOpFYwKNwIfWjnPzNuob));
        int tGsvtyAVkrDznETdweDC =
                (fOpFYwKNwIfWjnPzNuob < 1024) ? fOpFYwKNwIfWjnPzNuob : 1024;
        int
                KHClOltUSuqFVVErSxVb = (fOpFYwKNwIfWjnPzNuob + tGsvtyAVkrDznETdweDC -
                                        1) / tGsvtyAVkrDznETdweDC;
        MWSetDyForBackPropImpl<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(PVBPDNaynqYkBlDZgXgj, fOpFYwKNwIfWjnPzNuob);
    }
}

void
MWMaxPoolingLayerImpl::deallocate() {
    if (UdmcwaUkepxfZrpdpcAN) {
        CUDA_FREE_CALL(UdmcwaUkepxfZrpdpcAN);
        UdmcwaUkepxfZrpdpcAN =
                NULL;
    }
    if (PVBPDNaynqYkBlDZgXgj) {
        CUDA_FREE_CALL(PVBPDNaynqYkBlDZgXgj);
        PVBPDNaynqYkBlDZgXgj =
                NULL;
    }
}

void MWMaxPoolingLayerImpl::predict() {
    MWMaxPoolingLayer *
            maxpoolLayer = static_cast<MWMaxPoolingLayer *>(getLayer());
    MWTensorBase *
            ipTensorBase = maxpoolLayer->getInputTensor(0);
    MWTensorBase *opTensorBase =
            maxpoolLayer->getOutputTensor(0);
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    hipdnnTensorDescriptor_t XhAYHFyEVtlwoxGBuTpu =
            MWCNNLayerImpl::getCuDNNDescriptor(ipTensorBase);
    CUDNN_CALL(hipdnnPoolingForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                   muwRQxtWMMXAPxSuMYBw, getOnePtr(), XhAYHFyEVtlwoxGBuTpu, ipTensor->getData(),
                                   getZeroPtr(), *desc, opTensor->getData()));
    if (BUOdotSvmFyUWQKMUdra) {
        CUDNN_CALL(hipdnnPoolingBackward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                        muwRQxtWMMXAPxSuMYBw, getOnePtr(), *desc, opTensor->getData(), *desc,
                                        PVBPDNaynqYkBlDZgXgj, XhAYHFyEVtlwoxGBuTpu, ipTensor->getData(), getZeroPtr(),
                                        XhAYHFyEVtlwoxGBuTpu, UdmcwaUkepxfZrpdpcAN));
        int edQOkUJIZbwzEeIcCLzG =
                ipTensor->getNumElements();
        int tGsvtyAVkrDznETdweDC =
                (edQOkUJIZbwzEeIcCLzG < 1024) ? edQOkUJIZbwzEeIcCLzG : 1024;
        int
                KHClOltUSuqFVVErSxVb = (edQOkUJIZbwzEeIcCLzG + tGsvtyAVkrDznETdweDC -
                                        1) / tGsvtyAVkrDznETdweDC;
        doMWMaxPoolingLayerImpl<<<KHClOltUSuqFVVErSxVb,
        tGsvtyAVkrDznETdweDC>>>(UdmcwaUkepxfZrpdpcAN,
                                static_cast<MWTensor<float> *>(maxpoolLayer->getOutputTensor(1))->getData(),
                                edQOkUJIZbwzEeIcCLzG);
    }
    return;
}

void MWMaxPoolingLayerImpl::cleanup() {
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(muwRQxtWMMXAPxSuMYBw));
}

float *
MWMaxPoolingLayerImpl::getIndexData() {
    return
            static_cast<MWTensor<float> *>(getLayer()->getOutputTensor(1))->getData();
}

void __global__ __launch_bounds__(1024) MWSetDyForBackPropImpl(float *
PVBPDNaynqYkBlDZgXgj, const int fOpFYwKNwIfWjnPzNuob) {
    for (int i = blockDim.x *
                 blockIdx.x + threadIdx.x; i < fOpFYwKNwIfWjnPzNuob; i += blockDim.x * gridDim.x) {
        PVBPDNaynqYkBlDZgXgj[i] = i + 1;
    }
}

void __global__ __launch_bounds__(1024)
doMWMaxPoolingLayerImpl(float *UdmcwaUkepxfZrpdpcAN, float *
UWAGLbDcvybdWBtshhsr, const int BkwhtPQUCQKchmmimoXs) {
    for (int i = blockDim.x *
                 blockIdx.x + threadIdx.x; i < BkwhtPQUCQKchmmimoXs; i += blockDim.x * gridDim.x) {
        if
                (static_cast<int>(UdmcwaUkepxfZrpdpcAN[i]) != 0) {
            UWAGLbDcvybdWBtshhsr[static_cast<int>(UdmcwaUkepxfZrpdpcAN[i]) - 1] =
                    i;
        }
    }
}

MWFCLayerImpl::MWFCLayerImpl(MWCNNLayer *layer, MWTargetNetworkImpl *
ntwk_impl, int XNZmftADYzuZnIYIpBaT, int lteHjcLsItGbVPMQtGDB, const char *
xHViLEwTujGGrPZZgmbF, const char *JwxFdqOKggeawILBfGgg) :
        MWCNNLayerImpl(layer, ntwk_impl),
        CDJtexcMbXMWAmnNZsNf(XNZmftADYzuZnIYIpBaT),
        CGbFsczkgkhjcHoCKzBx(lteHjcLsItGbVPMQtGDB), vpXxoeEhdEosLSsYXkNG(NULL),
        wJyXsrUCMgxdIKVIJSyx(NULL), IwKnaBoXVubIRYcxEJLH(NULL),
        xHiBGayUfxIpXKkCTDNU(false) {
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF));
    createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
    CUDA_CALL(hipMalloc((void **) &vpXxoeEhdEosLSsYXkNG,
                         sizeof(float) * CDJtexcMbXMWAmnNZsNf * CGbFsczkgkhjcHoCKzBx));
    CUDA_CALL(hipMalloc((void **) &IwKnaBoXVubIRYcxEJLH,
                         sizeof(float) * CGbFsczkgkhjcHoCKzBx));
    wJyXsrUCMgxdIKVIJSyx =
            MALLOC_CALL(sizeof(float) * CDJtexcMbXMWAmnNZsNf * CGbFsczkgkhjcHoCKzBx);
    loadWeights(xHViLEwTujGGrPZZgmbF);
    loadBias(JwxFdqOKggeawILBfGgg);
}

MWFCLayerImpl::~MWFCLayerImpl() {}

void MWFCLayerImpl::propagateSize() {
    MWFCLayer *fcLayer = static_cast<MWFCLayer *>(getLayer());
    MWTensorBase *
            opTensor = fcLayer->getOutputTensor(0);
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    setDescriptor<float>(*desc, static_cast<MWTensor<float> *>(opTensor));
    if
            (opTensor->getSequenceLength() == 1) {
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, 1, CGbFsczkgkhjcHoCKzBx, 1, 1));
    } else {
        int dims[5]
                = {1, 1, CGbFsczkgkhjcHoCKzBx, 1, 1};
        int strides[5];
        MWTensorBase::getStrides(dims, 5, strides);
        CUDNN_CALL(hipdnnSetTensorNdDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_DATA_FLOAT, 5,
                                              dims, strides));
    }
}

void MWFCLayerImpl::loadWeights(const char *
QTXuPiGKeBUnmRzhlIDp) {
    FILE *QjgQHaUACFNSteMrRtRj =
            MWCNNLayer::openBinaryFile(QTXuPiGKeBUnmRzhlIDp);
    assert(QjgQHaUACFNSteMrRtRj);
    int
            dkLDkRwCBjeybwDHbKiE = CDJtexcMbXMWAmnNZsNf * CGbFsczkgkhjcHoCKzBx;
    call_fread(wJyXsrUCMgxdIKVIJSyx, sizeof(float), dkLDkRwCBjeybwDHbKiE,
               QjgQHaUACFNSteMrRtRj, QTXuPiGKeBUnmRzhlIDp);
    fclose(QjgQHaUACFNSteMrRtRj);
}

void
MWFCLayerImpl::prepareWeights() {
    if (!xHiBGayUfxIpXKkCTDNU) {
        int
                dkLDkRwCBjeybwDHbKiE = CDJtexcMbXMWAmnNZsNf * CGbFsczkgkhjcHoCKzBx;
        MWFCLayer *fcLayer = static_cast<MWFCLayer *>(getLayer());
        MWTensorBase *
                ipTensor = fcLayer->getInputTensor(0);
        if (ipTensor->getHeight() != 1 &&
            ipTensor->getWidth() != 1) {
            float *KZWeXiYFmdpQdsgidKeG =
                    MALLOC_CALL(sizeof(float) * ipTensor->getHeight() * ipTensor->getWidth());
            for (int
                         k = 0; k < dkLDkRwCBjeybwDHbKiE / ipTensor->getHeight() / ipTensor->getWidth(); k++) {
                for (int i = 0; i < ipTensor->getHeight() * ipTensor->getWidth(); i++)
                    KZWeXiYFmdpQdsgidKeG[i] = wJyXsrUCMgxdIKVIJSyx[k * ipTensor->getHeight() * ipTensor->getWidth() +
                                                                   i];
                for (int j = 0; j < ipTensor->getHeight(); j++)
                    for (int i = 0; i < ipTensor->getWidth();
                         i++)
                        wJyXsrUCMgxdIKVIJSyx[k * ipTensor->getHeight() * ipTensor->getWidth() +
                                             j * ipTensor->getWidth() + i] = KZWeXiYFmdpQdsgidKeG[j + i *
                                                                                                      ipTensor->getHeight()];
            }
            free(KZWeXiYFmdpQdsgidKeG);
        }
        CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG,
                             wJyXsrUCMgxdIKVIJSyx, sizeof(float) * dkLDkRwCBjeybwDHbKiE,
                             hipMemcpyHostToDevice));
        free(wJyXsrUCMgxdIKVIJSyx);
        wJyXsrUCMgxdIKVIJSyx = NULL;
        xHiBGayUfxIpXKkCTDNU = true;
    }
}

void
MWFCLayerImpl::loadBias(const char *QTXuPiGKeBUnmRzhlIDp) {
    MWFCLayer *fcLayer =
            static_cast<MWFCLayer *>(getLayer());
    MWTensorBase *opTensor =
            fcLayer->getOutputTensor(0);
    FILE *QjgQHaUACFNSteMrRtRj =
            MWCNNLayer::openBinaryFile(QTXuPiGKeBUnmRzhlIDp);
    assert(QjgQHaUACFNSteMrRtRj);
    int
            dkLDkRwCBjeybwDHbKiE = CGbFsczkgkhjcHoCKzBx;
    float *KZWeXiYFmdpQdsgidKeG =
            MALLOC_CALL(sizeof(float) * dkLDkRwCBjeybwDHbKiE);
    call_fread(KZWeXiYFmdpQdsgidKeG,
               sizeof(float), dkLDkRwCBjeybwDHbKiE, QjgQHaUACFNSteMrRtRj, QTXuPiGKeBUnmRzhlIDp);
    CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, KZWeXiYFmdpQdsgidKeG,
                         sizeof(float) * dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice));
    free(KZWeXiYFmdpQdsgidKeG);
    fclose(QjgQHaUACFNSteMrRtRj);
}

void
MWFCLayerImpl::postSetup() { prepareWeights(); }

void MWFCLayerImpl::predict() {
    MWFCLayer *fcLayer = static_cast<MWFCLayer *>(getLayer());
    MWTensorBase *
            ipTensorBase = fcLayer->getInputTensor(0);
    MWTensorBase *opTensorBase =
            fcLayer->getOutputTensor(0);
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    int numOutputRows =
            opTensor->getChannels();
    int numOutputCols =
            ipTensor->getBatchSize() * ipTensor->getSequenceLength();
    int innerDimension =
            ipTensor->getHeight() * ipTensor->getWidth() * ipTensor->getChannels();
    int
            URgvgDXnZskIYGdtimcU = 1;
    int UVzBVEOIylFjkSgHwFMp = 1;
    if (opTensor->getBatchSize() == 1 &&
        opTensor->getSequenceLength() == 1) {
        CUDA_CALL(hipMemcpy(opTensor->getData(),
                             IwKnaBoXVubIRYcxEJLH, sizeof(float) * numOutputRows, hipMemcpyDeviceToDevice));
        CUBLAS_CALL(hipblasSgemv(*dMxIKDGTITyhdLqIHBLA->getCublasHandle(), HIPBLAS_OP_T,
                                innerDimension, numOutputRows, getOnePtr(), vpXxoeEhdEosLSsYXkNG, innerDimension,
                                ipTensor->getData(), URgvgDXnZskIYGdtimcU, getOnePtr(), opTensor->getData(),
                                UVzBVEOIylFjkSgHwFMp));
    } else {
        CUBLAS_CALL(hipblasSgemm(*dMxIKDGTITyhdLqIHBLA->getCublasHandle(), HIPBLAS_OP_T,
                                HIPBLAS_OP_N, numOutputRows, numOutputCols, innerDimension, getOnePtr(),
                                vpXxoeEhdEosLSsYXkNG, innerDimension, ipTensor->getData(), innerDimension,
                                getZeroPtr(), opTensor->getData(), numOutputRows));
        hipdnnTensorDescriptor_t *
                desc = getDescriptor(opTensor->getSourcePortIndex());
        assert(desc);
        CUDNN_CALL(hipdnnAddTensor(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(), getOnePtr(),
                                  JgLfgHrHMEMmMYTettJF, IwKnaBoXVubIRYcxEJLH, getOnePtr(), *desc, opTensor->getData()));
    }
    return;
}

void MWFCLayerImpl::cleanup() {
    if (vpXxoeEhdEosLSsYXkNG) {
        CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG);
        vpXxoeEhdEosLSsYXkNG = NULL;
    }
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF));
    if
            (IwKnaBoXVubIRYcxEJLH) {
        CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH);
        IwKnaBoXVubIRYcxEJLH = NULL;
    }
}

MWSoftmaxLayerImpl::MWSoftmaxLayerImpl(MWCNNLayer *layer, MWTargetNetworkImpl *
ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) {
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&shEncNmxJsMuJKwbrwok));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&sjLjZacPSDNBEjAccrGU));
}

MWSoftmaxLayerImpl::~MWSoftmaxLayerImpl() {}

void
MWSoftmaxLayerImpl::propagateSize() {
    MWSoftmaxLayer *sfmxLayer =
            static_cast<MWSoftmaxLayer *>(getLayer());
    MWTensorBase *ipTensor =
            sfmxLayer->getInputTensor(0);
    MWTensorBase *opTensor =
            sfmxLayer->getOutputTensor(0);
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(shEncNmxJsMuJKwbrwok, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, ipTensor->getSequenceLength() * ipTensor->getBatchSize(),
                                          ipTensor->getChannels(), ipTensor->getHeight(), ipTensor->getWidth()));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(sjLjZacPSDNBEjAccrGU, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, opTensor->getSequenceLength() * opTensor->getBatchSize(),
                                          opTensor->getChannels(), opTensor->getHeight(), opTensor->getWidth()));
}

void
MWSoftmaxLayerImpl::predict() {
    MWSoftmaxLayer *sfmxLayer =
            static_cast<MWSoftmaxLayer *>(getLayer());
    MWTensorBase *ipTensorBase =
            sfmxLayer->getInputTensor(0);
    MWTensorBase *opTensorBase =
            sfmxLayer->getOutputTensor(0);
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    CUDNN_CALL(hipdnnSoftmaxForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                   HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_CHANNEL, getOnePtr(),
                                   shEncNmxJsMuJKwbrwok, ipTensor->getData(), getZeroPtr(),
                                   sjLjZacPSDNBEjAccrGU, opTensor->getData()));
}

void
MWSoftmaxLayerImpl::cleanup() {
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(shEncNmxJsMuJKwbrwok));
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(sjLjZacPSDNBEjAccrGU));
}

MWAvgPoolingLayerImpl::MWAvgPoolingLayerImpl(MWCNNLayer *layer,
                                             MWTargetNetworkImpl *ntwk_impl, int DqxLTLaJwwgQqmrtCDuu, int
                                             EvebzoroiuKkIxwjkGnD, int FshVHIJMRAhtQirYPlZd, int
                                             GDRXdUDklKFEYEfifhIH, int CpMjJjtGOeWOzwxpAAQP, int
                                             ClEhcJFlvGCgiavziIag, int DCdZnqpcBnvXVgEsLBnz, int
                                             DGzdAcREJHGXjyRzNjJV) : MWCNNLayerImpl(layer, ntwk_impl),
                                                                     TbrNrGxaFFHrzKUcfHNZ(NULL),
                                                                     DSsxcjIrUgZCKZovyNQf(DqxLTLaJwwgQqmrtCDuu),
                                                                     EfvWctmlsWAPsxXgdKWf(EvebzoroiuKkIxwjkGnD),
                                                                     DRzwhbNPpftRRIXXfHzd(DqxLTLaJwwgQqmrtCDuu),
                                                                     ECTnqgWHyHCHCLBZlffd(EvebzoroiuKkIxwjkGnD),
                                                                     FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd),
                                                                     FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH),
                                                                     CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP),
                                                                     CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag),
                                                                     CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz),
                                                                     CufLFODQDXTAPyRqYodN(DGzdAcREJHGXjyRzNjJV),
                                                                     IIiwAtyrOtLzLWAUlTey((CZNYmBcNFSZWvaCklqeM !=
                                                                                           CTCbzQMDaLxINPbODdng)
                                                                                          || (CqtPRJvHlGJFssiPzsOm !=
                                                                                              CufLFODQDXTAPyRqYodN)),
                                                                     nDsbARncmIrIaLubvLVZ(CpMjJjtGOeWOzwxpAAQP),
                                                                     nNULvWnBXnnWdpEkHPAH(DCdZnqpcBnvXVgEsLBnz) {
    CUDNN_CALL(hipdnnCreatePoolingDescriptor(&muwRQxtWMMXAPxSuMYBw));
    MWTensorBase *
            ipTensor = getLayer()->getInputTensor(0);
    if (IIiwAtyrOtLzLWAUlTey) {
        nDsbARncmIrIaLubvLVZ = 0;
        nNULvWnBXnnWdpEkHPAH = 0;
        TbrNrGxaFFHrzKUcfHNZ = new MWTensor<float>(-1, -1, -1, -1, -1, NULL, getLayer(), 0);
        if (!TbrNrGxaFFHrzKUcfHNZ) {
            MWCNNLayerImpl::throwAllocationError(__LINE__,
                                                 __FILE__);
        }
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&XhAYHFyEVtlwoxGBuTpu));
    } else { TbrNrGxaFFHrzKUcfHNZ = ipTensor; }
    assert(TbrNrGxaFFHrzKUcfHNZ != NULL);
    MWAvgPoolingLayer *avgpoolLayer = static_cast<MWAvgPoolingLayer *>(getLayer());
    MWTensorBase *opTensor = avgpoolLayer->getOutputTensor(0);
    createAndAddDescriptor(opTensor->getSourcePortIndex());
}

MWAvgPoolingLayerImpl::~MWAvgPoolingLayerImpl() {}

void
MWAvgPoolingLayerImpl::propagateSize() {
    MWTensorBase *ipTensor =
            getLayer()->getInputTensor(0);
    if ((DSsxcjIrUgZCKZovyNQf == -1) &&
        (EfvWctmlsWAPsxXgdKWf == -1)) {
        DRzwhbNPpftRRIXXfHzd = ipTensor->getHeight();
        ECTnqgWHyHCHCLBZlffd = ipTensor->getWidth();
    }
    int inputH;
    int inputW;
    if
            (IIiwAtyrOtLzLWAUlTey) {
        inputH = ipTensor->getHeight() +
                 CZNYmBcNFSZWvaCklqeM + CTCbzQMDaLxINPbODdng;
        inputW = ipTensor->getWidth() +
                 CqtPRJvHlGJFssiPzsOm + CufLFODQDXTAPyRqYodN;
    } else {
        inputH =
                ipTensor->getHeight();
        inputW = ipTensor->getWidth();
    }
    TbrNrGxaFFHrzKUcfHNZ->setHeight(inputH);
    TbrNrGxaFFHrzKUcfHNZ->setWidth(inputW);
    TbrNrGxaFFHrzKUcfHNZ->setChannels(ipTensor->getChannels());
    TbrNrGxaFFHrzKUcfHNZ->setBatchSize(ipTensor->getBatchSize());
    TbrNrGxaFFHrzKUcfHNZ->setSequenceLength(ipTensor->getSequenceLength());
    assert(TbrNrGxaFFHrzKUcfHNZ->getSequenceLength() == 1);
    if
            (IIiwAtyrOtLzLWAUlTey) {
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(XhAYHFyEVtlwoxGBuTpu, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, TbrNrGxaFFHrzKUcfHNZ->getBatchSize(),
                                              TbrNrGxaFFHrzKUcfHNZ->getChannels(),
                                              TbrNrGxaFFHrzKUcfHNZ->getHeight(), TbrNrGxaFFHrzKUcfHNZ->getWidth()));
    } else {
        XhAYHFyEVtlwoxGBuTpu = MWCNNLayerImpl::getCuDNNDescriptor(TbrNrGxaFFHrzKUcfHNZ);
    }
    MWTensorBase *opTensor = getLayer()->getOutputTensor(0);
    CUDNN_CALL(hipdnnSetPooling2dDescriptor(muwRQxtWMMXAPxSuMYBw,
                                           HIPDNN_POOLING_AVERAGE_COUNT_INCLUDE_PADDING, HIPDNN_NOT_PROPAGATE_NAN,
                                           DRzwhbNPpftRRIXXfHzd, ECTnqgWHyHCHCLBZlffd, nDsbARncmIrIaLubvLVZ,
                                           nNULvWnBXnnWdpEkHPAH, FrpxvsDMwwgbpqHXWxmN, FwLnexHgxHRquTKmNpoa));
    hipdnnTensorDescriptor_t *desc = getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    setDescriptor<float>(*desc,
                         static_cast<MWTensor<float> *>(opTensor));
}

void
MWAvgPoolingLayerImpl::allocate() {
    MWTensorBase *ipTensor =
            getLayer()->getInputTensor(0);
    if (IIiwAtyrOtLzLWAUlTey) {
        float *
                newInput;
        int inputH = ipTensor->getHeight() + CZNYmBcNFSZWvaCklqeM +
                     CTCbzQMDaLxINPbODdng;
        int inputW = ipTensor->getWidth() +
                     CqtPRJvHlGJFssiPzsOm + CufLFODQDXTAPyRqYodN;
        int paddedSize =
                ipTensor->getBatchSize() * ipTensor->getChannels() * inputH * inputW;
        CUDA_CALL(hipMalloc((void **) &newInput, sizeof(float) * paddedSize));
        CUDA_CALL(hipMemset(newInput, 0, sizeof(float) * paddedSize));
        static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->setData(newInput);
    }
}

void
MWAvgPoolingLayerImpl::deallocate() {
    if (TbrNrGxaFFHrzKUcfHNZ !=
        getLayer()->getInputTensor(0)) {
        assert(IIiwAtyrOtLzLWAUlTey);
        CUDA_FREE_CALL(static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->getData());
        static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->setData((float *) NULL);
    }
}

void
MWAvgPoolingLayerImpl::predict() {
    MWAvgPoolingLayer *avgpoolLayer =
            static_cast<MWAvgPoolingLayer *>(getLayer());
    MWTensorBase *opTensorBase =
            avgpoolLayer->getOutputTensor(0);
    MWTensorBase *ipTensorBase =
            avgpoolLayer->getInputTensor(0);
    MWTensor<float> *ipTensor =
            static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    if (TbrNrGxaFFHrzKUcfHNZ !=
        avgpoolLayer->getInputTensor()) {
        CUDA_CALL(hipMemset(static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->getData(),
                             0, sizeof(float) * TbrNrGxaFFHrzKUcfHNZ->getNumElements()));
        MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(),
                                 ipTensor->getWidth(), ipTensor->getChannels(), TbrNrGxaFFHrzKUcfHNZ->getHeight(),
                                 TbrNrGxaFFHrzKUcfHNZ->getWidth(), CZNYmBcNFSZWvaCklqeM, CqtPRJvHlGJFssiPzsOm,
                                 static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->getData(),
                                 ipTensor->getNumElements());
    }
    assert(opTensor->getData() !=
           static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->getData());
    hipdnnTensorDescriptor_t *desc = getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    CUDNN_CALL(hipdnnPoolingForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                   muwRQxtWMMXAPxSuMYBw, getOnePtr(), XhAYHFyEVtlwoxGBuTpu,
                                   static_cast<MWTensor<float> *>(TbrNrGxaFFHrzKUcfHNZ)->getData(), getZeroPtr(), *desc,
                                   opTensor->getData()));
}

void MWAvgPoolingLayerImpl::cleanup() {
    CUDNN_CALL(hipdnnDestroyPoolingDescriptor(muwRQxtWMMXAPxSuMYBw));
    if
            (TbrNrGxaFFHrzKUcfHNZ != getLayer()->getInputTensor(0)) {
        assert(IIiwAtyrOtLzLWAUlTey);
        CUDNN_CALL(hipdnnDestroyTensorDescriptor(XhAYHFyEVtlwoxGBuTpu));
    }
}

MWOutputLayerImpl::MWOutputLayerImpl(MWCNNLayer *layer, MWTargetNetworkImpl *
ntwk_impl) : MWCNNLayerImpl(layer, ntwk_impl) {}

MWOutputLayerImpl::~MWOutputLayerImpl() {}

void
MWOutputLayerImpl::propagateSize() {}

void
MWOutputLayerImpl::deallocateOutputData(int) {}

void
MWOutputLayerImpl::predict() {}

void MWOutputLayerImpl::cleanup() {}