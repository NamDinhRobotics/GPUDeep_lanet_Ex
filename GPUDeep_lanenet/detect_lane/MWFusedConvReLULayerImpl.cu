#include "MWFusedConvReLULayer.hpp"
#include "MWFusedConvReLULayerImpl.hpp"
#include <cassert>
#include <stdio.h>

MWFusedConvReLULayerImpl::MWFusedConvReLULayerImpl(MWCNNLayer *layer,
                                                   MWTargetNetworkImpl *ntwk_impl, int filt_H, int filt_W, int numGrps,
                                                   int
                                                   numChnls, int numFilts, int FshVHIJMRAhtQirYPlZd, int
                                                   GDRXdUDklKFEYEfifhIH, int CpMjJjtGOeWOzwxpAAQP, int
                                                   ClEhcJFlvGCgiavziIag, int DCdZnqpcBnvXVgEsLBnz, int
                                                   DGzdAcREJHGXjyRzNjJV, int AwZQzUhuWVLGrWgLHRuM, int
                                                   BHuHNDGoRwGRouCxeMbw, int eybNKlJCSDUvsznWynwK, const char *
xHViLEwTujGGrPZZgmbF, const char *JwxFdqOKggeawILBfGgg) :
        MWCNNLayerImpl(layer, ntwk_impl), vpXxoeEhdEosLSsYXkNG(NULL), IwKnaBoXVubIRYcxEJLH(NULL),
        UKtMXCCqdjeyaVHabkxg(NULL), XCLDbxHBtWRStETWIkId(NULL), HhKGcPZwrclEFnIdWerH(NULL),
        BLjrjqvCcCommiXWQLjs(filt_H), BRSPqxNffoBYKqpSVHne(filt_W),
        CCKWXUFWgrbBMjwfpOBN(numGrps), BlRIQPyqJZORKENzSdYf(numChnls),
        BuyZFXzwOMxcePIbCLfl(numFilts), FrpxvsDMwwgbpqHXWxmN(FshVHIJMRAhtQirYPlZd),
        FwLnexHgxHRquTKmNpoa(GDRXdUDklKFEYEfifhIH),
        CZNYmBcNFSZWvaCklqeM(CpMjJjtGOeWOzwxpAAQP),
        CTCbzQMDaLxINPbODdng(ClEhcJFlvGCgiavziIag),
        CqtPRJvHlGJFssiPzsOm(DCdZnqpcBnvXVgEsLBnz),
        CufLFODQDXTAPyRqYodN(DGzdAcREJHGXjyRzNjJV),
        AuqaQHxmPQSyYRemQvyX(AwZQzUhuWVLGrWgLHRuM),
        AzTsxYcYjIEJsGQbeYHm(BHuHNDGoRwGRouCxeMbw),
        fxxCPKTclxXPxrdMAkwi(eybNKlJCSDUvsznWynwK),
        IIiwAtyrOtLzLWAUlTey((CZNYmBcNFSZWvaCklqeM != CTCbzQMDaLxINPbODdng)
                             || (CqtPRJvHlGJFssiPzsOm != CufLFODQDXTAPyRqYodN)) {
#if (CUDNN_MAJOR < 6)
    throw std::runtime_error("Fused ConvReLU Layer only supported for cuDNN 6 or greater");
#else
    dMxIKDGTITyhdLqIHBLA = ntwk_impl;
    CUDNN_CALL(hipdnnCreateConvolutionDescriptor(&NXruhrCCiguRjAgSNDuz));
    CUDNN_CALL(hipdnnCreateFilterDescriptor(&QhTWatiCfcWYsHdkcyhZ));
    CUDNN_CALL(hipdnnCreateTensorDescriptor(&JgLfgHrHMEMmMYTettJF));
    CUDNN_CALL(hipdnnCreateActivationDescriptor(&oKIvzXXMucEDsTGGpdpm));
    MWTensorBase *ipTensor_conv = getLayer()->getInputTensor(0);
    int
            NZjOkZPwLzQsdEVkwMcX = CZNYmBcNFSZWvaCklqeM;
    int
            NbunkIVaMPVYgAQHXXYd = CqtPRJvHlGJFssiPzsOm;
    if
            (IIiwAtyrOtLzLWAUlTey) {
        NZjOkZPwLzQsdEVkwMcX = 0;
        NbunkIVaMPVYgAQHXXYd = 0;
        UKtMXCCqdjeyaVHabkxg = new MWTensor<float>(-1,
                                                   -1, -1, -1, -1, NULL, getLayer(), 0);
        if (!UKtMXCCqdjeyaVHabkxg) {
            MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__);
        }
        CUDNN_CALL(hipdnnCreateTensorDescriptor(&XhAYHFyEVtlwoxGBuTpu));
    } else {
        UKtMXCCqdjeyaVHabkxg = ipTensor_conv;
    }
    assert(UKtMXCCqdjeyaVHabkxg != NULL);
    bYBVtTnVUuGDUlaTmmHp = CZNYmBcNFSZWvaCklqeM;
    cQBKlCKXxecGPJrXBXdk =
            CqtPRJvHlGJFssiPzsOm;
    MWFusedConvReLULayer *fusedConvReluLayer =
            static_cast<MWFusedConvReLULayer *>(getLayer());
    CUDNN_CALL(hipdnnSetConvolution2dDescriptor(NXruhrCCiguRjAgSNDuz,
                                               NZjOkZPwLzQsdEVkwMcX, NbunkIVaMPVYgAQHXXYd, FrpxvsDMwwgbpqHXWxmN,
                                               FwLnexHgxHRquTKmNpoa, AuqaQHxmPQSyYRemQvyX, AzTsxYcYjIEJsGQbeYHm,
                                               HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));
#if (FP16_ENABLED == 1 && (CUDNN_MAJOR > 7 || (CUDNN_MAJOR == 7 && CUDNN_MINOR >= 2)))
    CUDNN_CALL(hipdnnSetConvolutionMathType(NXruhrCCiguRjAgSNDuz, CUDNN_TENSOR_OP_MATH_ALLOW_CONVERSION));
#endif
    if (CCKWXUFWgrbBMjwfpOBN > 1) {
        CUDNN_CALL(hipdnnSetConvolutionGroupCount(NXruhrCCiguRjAgSNDuz,
                                                 CCKWXUFWgrbBMjwfpOBN));
    }
    CUDNN_CALL(hipdnnSetActivationDescriptor(oKIvzXXMucEDsTGGpdpm,
                                            HIPDNN_ACTIVATION_RELU, HIPDNN_NOT_PROPAGATE_NAN, 0));
    int
            euppfEoiaoCTcVgRPVhA = BlRIQPyqJZORKENzSdYf * CCKWXUFWgrbBMjwfpOBN;
    int
            fSbUUBgjKRbNXrHrlOLo = BuyZFXzwOMxcePIbCLfl * CCKWXUFWgrbBMjwfpOBN;
    CUDNN_CALL(hipdnnSetFilter4dDescriptor(QhTWatiCfcWYsHdkcyhZ, HIPDNN_DATA_FLOAT,
                                          HIPDNN_TENSOR_NCHW, fSbUUBgjKRbNXrHrlOLo,
                                          euppfEoiaoCTcVgRPVhA / CCKWXUFWgrbBMjwfpOBN, BLjrjqvCcCommiXWQLjs,
                                          BRSPqxNffoBYKqpSVHne));
    CUDNN_CALL(hipdnnSetTensor4dDescriptor(JgLfgHrHMEMmMYTettJF, HIPDNN_TENSOR_NCHW,
                                          HIPDNN_DATA_FLOAT, 1, fSbUUBgjKRbNXrHrlOLo, 1, 1));
    int weightSize =
            BlRIQPyqJZORKENzSdYf * fSbUUBgjKRbNXrHrlOLo * BLjrjqvCcCommiXWQLjs * BRSPqxNffoBYKqpSVHne;
    CUDA_CALL(hipMalloc((void **) &vpXxoeEhdEosLSsYXkNG, sizeof(float) * weightSize));
    CUDA_CALL(hipMalloc((void **) &IwKnaBoXVubIRYcxEJLH,
                         sizeof(float) * fSbUUBgjKRbNXrHrlOLo));
    loadWeights(xHViLEwTujGGrPZZgmbF);
    loadBias(JwxFdqOKggeawILBfGgg);
    createAndAddDescriptor(getLayer()->getOutputTensor(0)->getSourcePortIndex());
#endif
}

MWFusedConvReLULayerImpl::~MWFusedConvReLULayerImpl() {}

void
MWFusedConvReLULayerImpl::propagateSize() {
#if (CUDNN_MAJOR >= 6)
    MWTensorBase *ipTensor_conv = getLayer()->getInputTensor(0);
    int inputH;
    int
            inputW;
    if (IIiwAtyrOtLzLWAUlTey) {
        inputH =
                ipTensor_conv->getHeight() + CZNYmBcNFSZWvaCklqeM + CTCbzQMDaLxINPbODdng;
        inputW = ipTensor_conv->getWidth() + CqtPRJvHlGJFssiPzsOm +
                 CufLFODQDXTAPyRqYodN;
    } else {
        inputH = ipTensor_conv->getHeight();
        inputW =
                ipTensor_conv->getWidth();
    }
    UKtMXCCqdjeyaVHabkxg->setHeight(inputH);
    UKtMXCCqdjeyaVHabkxg->setWidth(inputW);
    UKtMXCCqdjeyaVHabkxg->setChannels(ipTensor_conv->getChannels());
    UKtMXCCqdjeyaVHabkxg->setBatchSize(ipTensor_conv->getBatchSize());
    UKtMXCCqdjeyaVHabkxg->setSequenceLength(ipTensor_conv->getSequenceLength());
    assert(UKtMXCCqdjeyaVHabkxg->getSequenceLength() == 1);
    if
            (IIiwAtyrOtLzLWAUlTey) {
        CUDNN_CALL(hipdnnSetTensor4dDescriptor(XhAYHFyEVtlwoxGBuTpu, HIPDNN_TENSOR_NCHW,
                                              HIPDNN_DATA_FLOAT, UKtMXCCqdjeyaVHabkxg->getBatchSize(),
                                              UKtMXCCqdjeyaVHabkxg->getChannels(), UKtMXCCqdjeyaVHabkxg->getHeight(),
                                              UKtMXCCqdjeyaVHabkxg->getWidth()));
    } else {
        XhAYHFyEVtlwoxGBuTpu =
                MWCNNLayerImpl::getCuDNNDescriptor(UKtMXCCqdjeyaVHabkxg);
    }
    assert(BlRIQPyqJZORKENzSdYf ==
           UKtMXCCqdjeyaVHabkxg->getChannels() / CCKWXUFWgrbBMjwfpOBN);
    MWTensorBase *opTensor
            = getLayer()->getOutputTensor(0);
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
    setDescriptor<float>(*desc, static_cast<MWTensor<float> *>(opTensor));
#if (CUDNN_MAJOR < 7)
    {
   CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
   XhAYHFyEVtlwoxGBuTpu, QhTWatiCfcWYsHdkcyhZ, NXruhrCCiguRjAgSNDuz, *desc,
   HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &NMMfJylfQjiIUAKhXCJb)); }
#else
    {
        hipdnnConvolutionFwdAlgoPerf_t perf_results[3];
        int returnedAlgoCount;
        CUDNN_CALL(cudnnGetConvolutionForwardAlgorithm_v7(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                                          XhAYHFyEVtlwoxGBuTpu, QhTWatiCfcWYsHdkcyhZ,
                                                          NXruhrCCiguRjAgSNDuz, *desc, 3,
                                                          &returnedAlgoCount, perf_results));
        NMMfJylfQjiIUAKhXCJb = perf_results[0].algo;
    }
#endif
    if (CUDNN_VERSION < 7402) fixConvAlgo();
    size_t tnTPxeDjBsqLAPkJcPJX = 0;
    CUDNN_CALL(hipdnnGetConvolutionForwardWorkspaceSize(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                                       XhAYHFyEVtlwoxGBuTpu, QhTWatiCfcWYsHdkcyhZ, NXruhrCCiguRjAgSNDuz,
                                                       *desc,
                                                       NMMfJylfQjiIUAKhXCJb, &tnTPxeDjBsqLAPkJcPJX));
    if (tnTPxeDjBsqLAPkJcPJX >
        *dMxIKDGTITyhdLqIHBLA->getProposedWorkSpaceSize()) {
        dMxIKDGTITyhdLqIHBLA->setProposedWorkSpaceSize(tnTPxeDjBsqLAPkJcPJX);
    }
#endif
}

void MWFusedConvReLULayerImpl::allocate() {
    MWTensorBase *ipTensor_conv =
            getLayer()->getInputTensor(0);
    if (IIiwAtyrOtLzLWAUlTey) {
        float *
                newInput;
        int inputH = ipTensor_conv->getHeight() + CZNYmBcNFSZWvaCklqeM +
                     CTCbzQMDaLxINPbODdng;
        int inputW = ipTensor_conv->getWidth() +
                     CqtPRJvHlGJFssiPzsOm + CufLFODQDXTAPyRqYodN;
        int paddedSize =
                ipTensor_conv->getBatchSize() * ipTensor_conv->getChannels() * inputH * inputW;
        CUDA_CALL(hipMalloc((void **) &newInput, sizeof(float) * paddedSize));
        CUDA_CALL(hipMemset(newInput, 0, sizeof(float) * paddedSize));
        static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->setData(newInput);
    }
    XCLDbxHBtWRStETWIkId =
            static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData();
    setalpha2Ptr(getZeroPtr());
    int numInputs = getLayer()->getNumInputs();
    if
            (numInputs == 2) {
        setalpha2Ptr(getOnePtr());
        XCLDbxHBtWRStETWIkId =
                static_cast<MWTensor<float> *>(getLayer()->getInputTensor(1))->getData();
    }
}

void MWFusedConvReLULayerImpl::deallocate() {
    if (UKtMXCCqdjeyaVHabkxg !=
        getLayer()->getInputTensor(0)) {
        assert(IIiwAtyrOtLzLWAUlTey);
        CUDA_FREE_CALL(static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData());
        static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->setData((float *) NULL);
    }
}

void MWFusedConvReLULayerImpl::predict() {
    MWFusedConvReLULayer *
            fusedConvReluLayer = static_cast<MWFusedConvReLULayer *>(getLayer());
    MWTensorBase *ipTensorBase = fusedConvReluLayer->getInputTensor();
    MWTensorBase *opTensorBase = fusedConvReluLayer->getOutputTensor();
    MWTensor<float> *ipTensor = static_cast<MWTensor<float> *>(ipTensorBase);
    MWTensor<float> *opTensor = static_cast<MWTensor<float> *>(opTensorBase);
    if
            (UKtMXCCqdjeyaVHabkxg != fusedConvReluLayer->getInputTensor()) {
        CUDA_CALL(hipMemset(static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData(),
                             0, sizeof(float) * UKtMXCCqdjeyaVHabkxg->getNumElements()));
        MWCNNLayerImpl::padInput(ipTensor->getData(), ipTensor->getHeight(),
                                 ipTensor->getWidth(), ipTensor->getChannels(), UKtMXCCqdjeyaVHabkxg->getHeight(),
                                 UKtMXCCqdjeyaVHabkxg->getWidth(), bYBVtTnVUuGDUlaTmmHp, cQBKlCKXxecGPJrXBXdk,
                                 static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData(),
                                 ipTensor->getNumElements());
    }
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(opTensor->getSourcePortIndex());
    assert(desc);
#if (CUDNN_MAJOR >= 6)
    assert(opTensor->getData() !=
           static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData() ||
           (getLayer()->getNumInputs() == 2));
    CUDNN_CALL(cudnnConvolutionBiasActivationForward(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                                     getOnePtr(), XhAYHFyEVtlwoxGBuTpu,
                                                     static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData(),
                                                     QhTWatiCfcWYsHdkcyhZ, vpXxoeEhdEosLSsYXkNG, NXruhrCCiguRjAgSNDuz,
                                                     NMMfJylfQjiIUAKhXCJb,
                                                     dMxIKDGTITyhdLqIHBLA->getWorkSpace(),
                                                     *dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize(), getalpha2Ptr(),
                                                     *desc,
                                                     XCLDbxHBtWRStETWIkId, JgLfgHrHMEMmMYTettJF, IwKnaBoXVubIRYcxEJLH,
                                                     oKIvzXXMucEDsTGGpdpm,
                                                     *desc, opTensor->getData()));
#endif
}

void MWFusedConvReLULayerImpl::cleanup() {
    CUDNN_CALL(hipdnnDestroyConvolutionDescriptor(NXruhrCCiguRjAgSNDuz));
    CUDNN_CALL(hipdnnDestroyFilterDescriptor(QhTWatiCfcWYsHdkcyhZ));
    CUDNN_CALL(hipdnnDestroyActivationDescriptor(oKIvzXXMucEDsTGGpdpm));
    if
            (vpXxoeEhdEosLSsYXkNG) {
        CUDA_FREE_CALL(vpXxoeEhdEosLSsYXkNG);
        vpXxoeEhdEosLSsYXkNG = NULL;
    }
    CUDNN_CALL(hipdnnDestroyTensorDescriptor(JgLfgHrHMEMmMYTettJF));
    if
            (IwKnaBoXVubIRYcxEJLH) {
        CUDA_FREE_CALL(IwKnaBoXVubIRYcxEJLH);
        IwKnaBoXVubIRYcxEJLH = NULL;
    }
    if
            (UKtMXCCqdjeyaVHabkxg != getLayer()->getInputTensor(0)) {
        assert(IIiwAtyrOtLzLWAUlTey);
        CUDNN_CALL(hipdnnDestroyTensorDescriptor(XhAYHFyEVtlwoxGBuTpu));
    }
}

void
MWFusedConvReLULayerImpl::loadWeights(const char *QTXuPiGKeBUnmRzhlIDp) {
    FILE *
            QjgQHaUACFNSteMrRtRj = MWCNNLayer::openBinaryFile(QTXuPiGKeBUnmRzhlIDp);
    assert(QjgQHaUACFNSteMrRtRj);
    int dkLDkRwCBjeybwDHbKiE =
            BlRIQPyqJZORKENzSdYf * CCKWXUFWgrbBMjwfpOBN * BuyZFXzwOMxcePIbCLfl * BLjrjqvCcCommiXWQLjs *
            BRSPqxNffoBYKqpSVHne;
    float *KZWeXiYFmdpQdsgidKeG = MALLOC_CALL(sizeof(float) * dkLDkRwCBjeybwDHbKiE);
    call_fread(KZWeXiYFmdpQdsgidKeG, sizeof(float), dkLDkRwCBjeybwDHbKiE, QjgQHaUACFNSteMrRtRj,
               QTXuPiGKeBUnmRzhlIDp);
    CUDA_CALL(hipMemcpy(vpXxoeEhdEosLSsYXkNG, KZWeXiYFmdpQdsgidKeG,
                         sizeof(float) * dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice));
#if 0
    printf("%s loaded. Size = %d. %f\n", QTXuPiGKeBUnmRzhlIDp, dkLDkRwCBjeybwDHbKiE, KZWeXiYFmdpQdsgidKeG[0]);
#endif
    free(KZWeXiYFmdpQdsgidKeG);
    fclose(QjgQHaUACFNSteMrRtRj);
    return;
}

void
MWFusedConvReLULayerImpl::loadBias(const char *QTXuPiGKeBUnmRzhlIDp) {
    FILE *
            QjgQHaUACFNSteMrRtRj = MWCNNLayer::openBinaryFile(QTXuPiGKeBUnmRzhlIDp);
    assert(QjgQHaUACFNSteMrRtRj);
    int dkLDkRwCBjeybwDHbKiE =
            CCKWXUFWgrbBMjwfpOBN * BuyZFXzwOMxcePIbCLfl;
    float *KZWeXiYFmdpQdsgidKeG =
            MALLOC_CALL(sizeof(float) * dkLDkRwCBjeybwDHbKiE);
    call_fread(KZWeXiYFmdpQdsgidKeG,
               sizeof(float), dkLDkRwCBjeybwDHbKiE, QjgQHaUACFNSteMrRtRj, QTXuPiGKeBUnmRzhlIDp);
    CUDA_CALL(hipMemcpy(IwKnaBoXVubIRYcxEJLH, KZWeXiYFmdpQdsgidKeG,
                         sizeof(float) * dkLDkRwCBjeybwDHbKiE, hipMemcpyHostToDevice));
    free(KZWeXiYFmdpQdsgidKeG);
    fclose(QjgQHaUACFNSteMrRtRj);
    return;
}

void
MWFusedConvReLULayerImpl::postSetup() {
    if (dMxIKDGTITyhdLqIHBLA->getAutoTune()) { getConvAlgoTuned(); } else { getConvAlgoWorkSpaceLimit(); }
}

void
MWFusedConvReLULayerImpl::getConvAlgoTuned() {
    MWTensorBase *opTensorBase =
            getLayer()->getOutputTensor(0);
    MWTensor<float> *opTensor =
            static_cast<MWTensor<float> *>(opTensorBase);
    hipdnnConvolutionFwdAlgoPerf_t
            perf_results[3];
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex());
    assert(desc);
    int returnedAlgoCount;
    CUDNN_CALL(hipdnnFindConvolutionForwardAlgorithmEx(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                                      XhAYHFyEVtlwoxGBuTpu,
                                                      static_cast<MWTensor<float> *>(UKtMXCCqdjeyaVHabkxg)->getData(),
                                                      QhTWatiCfcWYsHdkcyhZ, vpXxoeEhdEosLSsYXkNG, NXruhrCCiguRjAgSNDuz,
                                                      *desc,
                                                      opTensor->getData(), 3, &returnedAlgoCount, &perf_results[0],
                                                      dMxIKDGTITyhdLqIHBLA->getWorkSpace(),
                                                      *dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize()));
    NMMfJylfQjiIUAKhXCJb =
            perf_results[0].algo;
    if (CUDNN_VERSION < 7402) fixConvAlgo();
}

void
MWFusedConvReLULayerImpl::getConvAlgoWorkSpaceLimit() {
    hipdnnTensorDescriptor_t *desc =
            getDescriptor(getLayer()->getOutputTensor()->getSourcePortIndex());
    assert(desc);
    CUDNN_CALL(hipdnnGetConvolutionForwardAlgorithm(*dMxIKDGTITyhdLqIHBLA->getCudnnHandle(),
                                                   XhAYHFyEVtlwoxGBuTpu, QhTWatiCfcWYsHdkcyhZ, NXruhrCCiguRjAgSNDuz,
                                                   *desc,
                                                   HIPDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
                                                   *dMxIKDGTITyhdLqIHBLA->getAllocatedWorkSpaceSize(),
                                                   &NMMfJylfQjiIUAKhXCJb));
    if
            (CUDNN_VERSION < 7402)
        fixConvAlgo();
}

void
MWFusedConvReLULayerImpl::fixConvAlgo() {
    int inputH =
            UKtMXCCqdjeyaVHabkxg->getHeight();
    int inputW = UKtMXCCqdjeyaVHabkxg->getWidth();
    if (NMMfJylfQjiIUAKhXCJb == HIPDNN_CONVOLUTION_FWD_ALGO_FFT && (inputH > 64 ||
                                                                   inputW > 64)) {
        NMMfJylfQjiIUAKhXCJb = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
    }
}