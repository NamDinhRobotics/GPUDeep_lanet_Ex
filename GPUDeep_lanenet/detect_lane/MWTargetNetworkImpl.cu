#include "MWTargetNetworkImpl.hpp"
#include "MWCNNLayerImpl.hpp"
#include <cassert>
#include <cmath>
#include <algorithm>

void MWTargetNetworkImpl::allocate(int BufSize, int numBufsToAlloc) {
    numBufs
            = numBufsToAlloc;
    for (int i = 0; i < numBufs; i++) {
        float *memPtr = 0;
        CUDA_CALL(hipMalloc((void **) &memPtr, sizeof(float) * BufSize));
        memBuffer.push_back(memPtr);
    }
}

void
MWTargetNetworkImpl::allocatePermuteBuffers(int bufSize, int numBufsToAlloc) {
    for (int i = 0; i < numBufsToAlloc; i++) {
        float *memPtr = 0;
        CUDA_CALL(hipMalloc((void **) &memPtr, sizeof(float) * bufSize));
        mtolGPkUMBYDlSSqrRzc.push_back(memPtr);
    }
}

void
MWTargetNetworkImpl::preSetup() {
    NldNILHvuQqQPSAHXxdT = new
            hipblasHandle_t;
    if (!NldNILHvuQqQPSAHXxdT) {
        MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__);
    }
    hipblasCreate(NldNILHvuQqQPSAHXxdT);
    NmExSIssnXpisMKKatUq = new
            hipdnnHandle_t;
    if (!NmExSIssnXpisMKKatUq) {
        MWCNNLayerImpl::throwAllocationError(__LINE__, __FILE__);
    }
    hipdnnCreate(NmExSIssnXpisMKKatUq);
}

void
MWTargetNetworkImpl::postSetup(MWCNNLayer *layers[], int numLayers) {
    if
            (*getProposedWorkSpaceSize() > *getAllocatedWorkSpaceSize()) {
        if
                (xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); }
        createWorkSpace(xcusoQxPPodcHwVviCWI);
        while ((!xcusoQxPPodcHwVviCWI) &&
               (*getProposedWorkSpaceSize() > 0)) {
            setProposedWorkSpaceSize(MWTargetNetworkImpl::getNextProposedWorkSpaceSize(*getProposedWorkSpaceSize()));
            createWorkSpace(xcusoQxPPodcHwVviCWI);
        }
    }
    for (int i = 0; i < numLayers; i++) { layers[i]->postSetup(); }
}

size_t
MWTargetNetworkImpl::getNextProposedWorkSpaceSize(size_t failedWorkSpaceSize) {
    assert(failedWorkSpaceSize > 0);
    return failedWorkSpaceSize / 2;
}

void
MWTargetNetworkImpl::createWorkSpace(float *&xkUNToJIgvoLoUQuzKRF) {
    hipError_t rlQsibXJSWJVnUVpdNeL = hipMalloc((void **) &xkUNToJIgvoLoUQuzKRF,
                                                  *getProposedWorkSpaceSize());
    if (rlQsibXJSWJVnUVpdNeL != hipSuccess) {
        xkUNToJIgvoLoUQuzKRF = NULL;
        setAllocatedWorkSpaceSize(0);
        rlQsibXJSWJVnUVpdNeL = hipGetLastError();
    } else {
        setAllocatedWorkSpaceSize(*getProposedWorkSpaceSize());
    }
}

void
MWTargetNetworkImpl::destroyWorkSpace(float *&xkUNToJIgvoLoUQuzKRF) {
    CUDA_FREE_CALL(xkUNToJIgvoLoUQuzKRF);
    xkUNToJIgvoLoUQuzKRF = NULL;
    setAllocatedWorkSpaceSize(0);
}

void
MWTargetNetworkImpl::setProposedWorkSpaceSize(size_t wss) {
    ncMionCCOTOYjWcmaIVD = wss;
}

size_t *
MWTargetNetworkImpl::getProposedWorkSpaceSize() {
    return
            &ncMionCCOTOYjWcmaIVD;
}

void
MWTargetNetworkImpl::setAllocatedWorkSpaceSize(size_t wss) {
    GvDXGhRLfipwBoRPoGfI = wss;
}

size_t *
MWTargetNetworkImpl::getAllocatedWorkSpaceSize() {
    return
            &GvDXGhRLfipwBoRPoGfI;
}

float *
MWTargetNetworkImpl::getWorkSpace() { return xcusoQxPPodcHwVviCWI; }

float *
MWTargetNetworkImpl::getPermuteBuffer(int bufIndex) {
    return
            mtolGPkUMBYDlSSqrRzc[bufIndex];
}

hipblasHandle_t *
MWTargetNetworkImpl::getCublasHandle() { return NldNILHvuQqQPSAHXxdT; }

hipdnnHandle_t *MWTargetNetworkImpl::getCudnnHandle() {
    return
            NmExSIssnXpisMKKatUq;
}

void MWTargetNetworkImpl::setAutoTune(bool
                                      autotune) { MW_autoTune = autotune; }

bool MWTargetNetworkImpl::getAutoTune()
const { return MW_autoTune; }

void MWTargetNetworkImpl::deallocate() {
    for (int
                 i = 0; i < memBuffer.size(); i++) {
        float *memPtr = memBuffer[i];
        if (memPtr) {
            CUDA_FREE_CALL(memPtr);
        }
    }
    memBuffer.clear();
    for (int i = 0; i <
                    mtolGPkUMBYDlSSqrRzc.size(); i++) {
        float *memPtr =
                mtolGPkUMBYDlSSqrRzc[i];
        if (memPtr) { CUDA_FREE_CALL(memPtr); }
    }
    mtolGPkUMBYDlSSqrRzc.clear();
}

void MWTargetNetworkImpl::cleanup() {
    if
            (xcusoQxPPodcHwVviCWI) { destroyWorkSpace(xcusoQxPPodcHwVviCWI); }
    if
            (NldNILHvuQqQPSAHXxdT) {
        hipblasDestroy(*NldNILHvuQqQPSAHXxdT);
        delete
                NldNILHvuQqQPSAHXxdT;
    }
    if (NmExSIssnXpisMKKatUq) {
        hipdnnDestroy(*NmExSIssnXpisMKKatUq);
        delete NmExSIssnXpisMKKatUq;
    }
}

float *MWTargetNetworkImpl::getBufferPtr(int bufferIndex) {
    return
            memBuffer[bufferIndex];
}

MWTargetNetworkImpl::~MWTargetNetworkImpl() {}