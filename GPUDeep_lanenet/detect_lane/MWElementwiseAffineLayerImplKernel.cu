#include "hip/hip_runtime.h"
#include "MWKernelHeaders.hpp"
#include <math.h>
#include <stdio.h>

void __global__ __launch_bounds__(1024) scale_scalar_kernel(float *
inputBuffer, float *outputBuffer, float *pvpNsgGssdTxeVoFIkXI, long int
                                                            YNDVziqpDddiXQKYZZhX) {
    for (long int idx = blockDim.x * blockIdx.x +
                        threadIdx.x; idx < YNDVziqpDddiXQKYZZhX; idx += blockDim.x * gridDim.x) {
        outputBuffer[idx] = pvpNsgGssdTxeVoFIkXI[0] * inputBuffer[idx];
    }
}

void __global__
__launch_bounds__(1024) scale_vector_kernel(float *inputBuffer, float *
outputBuffer, float *pvpNsgGssdTxeVoFIkXI, double YOWMnLKOMqAODXiVNoGy,
                                            double YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX) {
    for
            (long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx <
                                                                   YNDVziqpDddiXQKYZZhX; idx += blockDim.x *
                                                                                                gridDim.x) {
        double batchIdx =
                floor(idx / YNmJhGSUszJKxsodxiuV);
        double i_batch = idx - (batchIdx *
                                YNmJhGSUszJKxsodxiuV);
        double channelIdx = floor(i_batch /
                                  YOWMnLKOMqAODXiVNoGy);
        outputBuffer[idx] =
                pvpNsgGssdTxeVoFIkXI[static_cast<long int>(channelIdx)] * inputBuffer[idx];
    }
}

void
__global__ __launch_bounds__(1024) scale_matrix2d_kernel(float *inputBuffer,
                                                         float *outputBuffer, float *pvpNsgGssdTxeVoFIkXI, double
                                                         YOWMnLKOMqAODXiVNoGy, long int YNDVziqpDddiXQKYZZhX) {
    for (long int
                 idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YNDVziqpDddiXQKYZZhX; idx +=
                                                                                                  blockDim.x *
                                                                                                  gridDim.x) {
        double totalChannelIdx = floor(idx /
                                       YOWMnLKOMqAODXiVNoGy);
        double i_channel = idx - (totalChannelIdx *
                                  YOWMnLKOMqAODXiVNoGy);
        outputBuffer[idx] =
                pvpNsgGssdTxeVoFIkXI[static_cast<long int>(i_channel)] * inputBuffer[idx];
    }
}

void
__global__ __launch_bounds__(1024) scale_tensor3d_kernel(float *inputBuffer,
                                                         float *outputBuffer, float *pvpNsgGssdTxeVoFIkXI, double
                                                         YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX) {
    for (long int
                 idx = blockDim.x * blockIdx.x + threadIdx.x; idx < YNDVziqpDddiXQKYZZhX; idx +=
                                                                                                  blockDim.x *
                                                                                                  gridDim.x) {
        double batchIdx = floor(idx /
                                YNmJhGSUszJKxsodxiuV);
        double i_batch = idx - (batchIdx *
                                YNmJhGSUszJKxsodxiuV);
        outputBuffer[idx] =
                pvpNsgGssdTxeVoFIkXI[static_cast<long int>(i_batch)] * inputBuffer[idx];
    }
}

void
__global__ __launch_bounds__(1024) offset_scalar_kernel(float *inputBuffer,
                                                        float *outputBuffer, float *gNROjwaqhxDPvBWUCUcQ,
                                                        long int YNDVziqpDddiXQKYZZhX,
                                                        bool ZUTPCvgISoRdtnhGqXzM, int bQjijJlpNAVdwDDQgpaX, int
                                                        veFyKKHbdqBIvQLYBqfF) {
    for (long int idx = blockDim.x * blockIdx.x +
                        threadIdx.x; idx < YNDVziqpDddiXQKYZZhX; idx += blockDim.x * gridDim.x) {
        float
                out = inputBuffer[idx] + gNROjwaqhxDPvBWUCUcQ[0];
        if (ZUTPCvgISoRdtnhGqXzM) {
            out =
                    out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out;
            out = out <
                  bQjijJlpNAVdwDDQgpaX ? bQjijJlpNAVdwDDQgpaX : out;
        }
        outputBuffer[idx] = out;
    }
}

void __global__ __launch_bounds__(1024) offset_vector_kernel(float *
inputBuffer, float *outputBuffer, float *gNROjwaqhxDPvBWUCUcQ, double
                                                             YOWMnLKOMqAODXiVNoGy, double YNmJhGSUszJKxsodxiuV, long int
                                                             YNDVziqpDddiXQKYZZhX, bool ZUTPCvgISoRdtnhGqXzM,
                                                             int bQjijJlpNAVdwDDQgpaX, int
                                                             veFyKKHbdqBIvQLYBqfF) {
    for (long int idx = blockDim.x * blockIdx.x +
                        threadIdx.x; idx < YNDVziqpDddiXQKYZZhX; idx += blockDim.x * gridDim.x) {
        double batchIdx = floor(idx / YNmJhGSUszJKxsodxiuV);
        double i_batch =
                idx - (batchIdx * YNmJhGSUszJKxsodxiuV);
        double channelIdx =
                floor(i_batch / YOWMnLKOMqAODXiVNoGy);
        float out = inputBuffer[idx] +
                    gNROjwaqhxDPvBWUCUcQ[static_cast<long int>(channelIdx)];
        if
                (ZUTPCvgISoRdtnhGqXzM) {
            out = out > veFyKKHbdqBIvQLYBqfF ?
                  veFyKKHbdqBIvQLYBqfF : out;
            out = out < bQjijJlpNAVdwDDQgpaX ?
                  bQjijJlpNAVdwDDQgpaX : out;
        }
        outputBuffer[idx] = out;
    }
}

void __global__
__launch_bounds__(1024) offset_matrix2d_kernel(float *inputBuffer, float *
outputBuffer, float *gNROjwaqhxDPvBWUCUcQ, double YOWMnLKOMqAODXiVNoGy,
                                               long int YNDVziqpDddiXQKYZZhX, bool ZUTPCvgISoRdtnhGqXzM, int
                                               bQjijJlpNAVdwDDQgpaX, int veFyKKHbdqBIvQLYBqfF) {
    for (long int idx =
            blockDim.x * blockIdx.x + threadIdx.x; idx < YNDVziqpDddiXQKYZZhX; idx +=
                                                                                       blockDim.x * gridDim.x) {
        double totalChannelIdx = floor(idx /
                                       YOWMnLKOMqAODXiVNoGy);
        double i_channel = idx - (totalChannelIdx *
                                  YOWMnLKOMqAODXiVNoGy);
        float out = inputBuffer[idx] +
                    gNROjwaqhxDPvBWUCUcQ[static_cast<long int>(i_channel)];
        if (ZUTPCvgISoRdtnhGqXzM) {
            out = out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out;
            out = out <
                  bQjijJlpNAVdwDDQgpaX ? bQjijJlpNAVdwDDQgpaX : out;
        }
        outputBuffer[idx] = out;
    }
}

void __global__ __launch_bounds__(1024) offset_tensor3d_kernel(float *
inputBuffer, float *outputBuffer, float *gNROjwaqhxDPvBWUCUcQ, double
                                                               YNmJhGSUszJKxsodxiuV, long int YNDVziqpDddiXQKYZZhX, bool
                                                               ZUTPCvgISoRdtnhGqXzM, int bQjijJlpNAVdwDDQgpaX,
                                                               int veFyKKHbdqBIvQLYBqfF) {
    for (long int idx = blockDim.x * blockIdx.x + threadIdx.x; idx <
                                                               YNDVziqpDddiXQKYZZhX; idx += blockDim.x * gridDim.x) {
        double batchIdx =
                floor(idx / YNmJhGSUszJKxsodxiuV);
        double i_batch = idx - (batchIdx *
                                YNmJhGSUszJKxsodxiuV);
        float out = inputBuffer[idx] +
                    gNROjwaqhxDPvBWUCUcQ[static_cast<long int>(i_batch)];
        if (ZUTPCvgISoRdtnhGqXzM) {
            out = out > veFyKKHbdqBIvQLYBqfF ? veFyKKHbdqBIvQLYBqfF : out;
            out = out <
                  bQjijJlpNAVdwDDQgpaX ? bQjijJlpNAVdwDDQgpaX : out;
        }
        outputBuffer[idx] = out;
    }
}