#include "hip/hip_runtime.h"
//
// File: predict.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 15-Dec-2020 12:44:50
//

// Include Files
#include "predict.h"
#include "DeepLearningNetwork.h"
#include "detect_lane_internal_types.h"
#include "MWCudaDimUtility.hpp"
#include "cnn_api.hpp"

// Type Definitions
struct cell_wrap_6 {
    float f1[6];
};

struct cell_wrap_3 {
    float f1[154587];
};

// Function Declarations
static __global__ void DeepLearningNetwork_predict_kernel30(const float
                                                            varargin_1[154587], cell_wrap_3 miniBatchT[1]);

static __global__ void DeepLearningNetwork_predict_kernel31(const cell_wrap_6
                                                            outputsMiniBatch[1], float varargout_1[6]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float varargin_1[154587]
//                cell_wrap_3 miniBatchT[1]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void
DeepLearningNetwork_predict_kernel30(const float varargin_1[154587],
                                     cell_wrap_3 miniBatchT[1]) {
    unsigned long threadId;
    int i;
    int i1;
    int p;
    threadId = mwGetGlobalThreadIndex();
    i = static_cast<int>(threadId % 227UL);
    threadId = (threadId - static_cast<unsigned long>(i)) / 227UL;
    i1 = static_cast<int>(threadId % 227UL);
    threadId = (threadId - static_cast<unsigned long>(i1)) / 227UL;
    p = static_cast<int>(threadId);
    if ((static_cast<int>((static_cast<int>(p < 3)) && (static_cast<int>(i1 < 227))))
        && (static_cast<int>(i < 227))) {
        miniBatchT[0].f1[(i + 227 * i1) + 51529 * p] = varargin_1[(i1 + 227 * i) +
                                                                  51529 * p];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const cell_wrap_6 outputsMiniBatch[1]
//                float varargout_1[6]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void
DeepLearningNetwork_predict_kernel31(const cell_wrap_6 outputsMiniBatch[1],
                                     float varargout_1[6]) {
    int i;
    i = static_cast<int>(mwGetGlobalThreadIndex());
    if (i < 6) {
        varargout_1[i] = outputsMiniBatch[0].f1[i];
    }
}

//
// Arguments    : lanenet0_0 *obj
//                const float varargin_1[154587]
//                float varargout_1[6]
// Return Type  : void
//
namespace coder {
    void DeepLearningNetwork_predict(lanenet0_0 *obj, const float varargin_1
    [154587], float varargout_1[6]) {
        cell_wrap_3 (*gpu_miniBatchT)[1];
        cell_wrap_6 (*gpu_outputsMiniBatch)[1];
        float (*gpu_varargin_1)[154587];
        float (*gpu_varargout_1)[6];
        hipMalloc(&gpu_varargout_1, 24UL);
        hipMalloc(&gpu_outputsMiniBatch, 24UL);
        hipMalloc(&gpu_miniBatchT, 618348UL);
        hipMalloc(&gpu_varargin_1, 618348UL);
        hipMemcpy(gpu_varargin_1, (void *) &varargin_1[0], 618348UL,
                   hipMemcpyHostToDevice);
        DeepLearningNetwork_predict_kernel30<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>
                (*gpu_varargin_1, *gpu_miniBatchT);
        hipMemcpy(obj->getInputDataPointer(0), (*gpu_miniBatchT)[0].f1, obj->
                           layers[0]->getOutputTensor(0)->getNumElements() * sizeof(float),
                   hipMemcpyDeviceToDevice);
        obj->predict();
        hipMemcpy((*gpu_outputsMiniBatch)[0].f1, obj->getLayerOutput(17, 0),
                   obj->layers[17]->getOutputTensor(0)->getNumElements() * sizeof
                           (float), hipMemcpyDeviceToDevice);
        DeepLearningNetwork_predict_kernel31<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
                (*gpu_outputsMiniBatch, *gpu_varargout_1);
        hipMemcpy(&varargout_1[0], gpu_varargout_1, 24UL, hipMemcpyDeviceToHost);
        hipFree(*gpu_varargin_1);
        hipFree(*gpu_miniBatchT);
        hipFree(*gpu_outputsMiniBatch);
        hipFree(*gpu_varargout_1);
    }
}

//
// File trailer for predict.cu
//
// [EOF]
//
