#include "hip/hip_runtime.h"
//
// File: detect_lane.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 15-Dec-2020 12:44:50
//

// Include Files
#include "detect_lane.h"
#include "DeepLearningNetwork.h"
#include "detect_lane_data.h"
#include "detect_lane_initialize.h"
#include "detect_lane_internal_types.h"
#include "predict.h"
#include "MWCudaDimUtility.hpp"
#include <cmath>

// Variable Definitions
static lanenet0_0 lanenet;
static boolean_T lanenet_not_empty;

// Function Declarations
static __global__ void detect_lane_kernel1(const float frame[154587], float b
[154587]);

static __global__ void detect_lane_kernel10(double b_x[9]);

static __global__ void detect_lane_kernel11(const double t1, const double t2,
                                            double Tinv[9]);

static __global__ void detect_lane_kernel12(const double t1, const double t2,
                                            const int p2, double Tinv[9]);

static __global__ void detect_lane_kernel13(const double t1, const double t2,
                                            const int p3, double Tinv[9]);

static __global__ void detect_lane_kernel14(const float lt_y[28], float fv[84]);

static __global__ void detect_lane_kernel15(const double Tinv[9], const float
fv[84], float U[84]);

static __global__ void detect_lane_kernel16(const float U[84], float b[56]);

static __global__ void detect_lane_kernel17(const float b[56], float U[84]);

static __global__ void detect_lane_kernel18(const float U[84], float ltPts[56]);

static __global__ void detect_lane_kernel19(const double Tinv[9], double b_x[9]);

static __global__ void detect_lane_kernel2(const double laneCoeffMeans[6], const
double laneCoeffStds[6], float lanecoeffsNetworkOutput[6]);

static __global__ void detect_lane_kernel20(double b_x[9]);

static __global__ void detect_lane_kernel21(double b_x[9]);

static __global__ void detect_lane_kernel22(const double t1, const double t2,
                                            double Tinv[9]);

static __global__ void detect_lane_kernel23(const double t1, const double t2,
                                            const int p2, double Tinv[9]);

static __global__ void detect_lane_kernel24(const double t1, const double t2,
                                            const int p3, double Tinv[9]);

static __global__ void detect_lane_kernel25(const float rt_y[28], float fv[84]);

static __global__ void detect_lane_kernel26(const double Tinv[9], const float
fv[84], float U[84]);

static __global__ void detect_lane_kernel27(const float U[84], float b[56]);

static __global__ void detect_lane_kernel28(const float b[56], float U[84]);

static __global__ void detect_lane_kernel29(const float U[84], float rtPts[56]);

static __global__ void detect_lane_kernel3(const float lanecoeffsNetworkOutput[6],
                                           float rt_y[28]);

static __global__ void detect_lane_kernel4(const float lanecoeffsNetworkOutput,
                                           float rt_y[28]);

static __global__ void detect_lane_kernel5(const float lanecoeffsNetworkOutput[6],
                                           float lt_y[28]);

static __global__ void detect_lane_kernel6(const float lanecoeffsNetworkOutput,
                                           float lt_y[28]);

static __global__ void detect_lane_kernel7(double Tinv[9]);

static __global__ void detect_lane_kernel8(const double Tinv[9], double b_x[9]);

static __global__ void detect_lane_kernel9(double b_x[9]);

// Function Definitions
//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float frame[154587]
//                float b[154587]
// Return Type  : void
//
static __global__ __launch_bounds__(512, 1) void detect_lane_kernel1(const float
                                                                     frame[154587], float b[154587]) {
    unsigned long threadId;
    int ibcol;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    jtilecol = static_cast<int>(threadId % 227UL);
    threadId = (threadId - static_cast<unsigned long>(jtilecol)) / 227UL;
    ibcol = static_cast<int>(threadId % 227UL);
    threadId = (threadId - static_cast<unsigned long>(ibcol)) / 227UL;
    k = static_cast<int>(threadId);
    if ((static_cast<int>((static_cast<int>(k < 3)) && (static_cast<int>(ibcol <
                                                                         227)))) &&
        (static_cast<int>(jtilecol < 227))) {
        b[(ibcol + 227 * jtilecol) + 51529 * k] = frame[(jtilecol + 227 * ibcol) +
                                                        51529 * k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double b_x[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel10(double b_x
[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        b_x[4] -= b_x[1] * 1.0305949982581226;
        b_x[5] = -0.0032316839464807288 - b_x[2] * 1.0305949982581226;
        b_x[7] -= b_x[1] * -0.22205377950113064;
        b_x[8] = 1.9916790026632809E-35 - b_x[2] * -0.22205377950113064;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double t1
//                const double t2
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel11(const
                                                                     double t1, const double t2, double Tinv[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[6] = ((1.0 - 1.0305949982581226 * t2) - -0.22205377950113064 * t1) /
                  1.1512965678044422;
        Tinv[7] = t2;
        Tinv[8] = t1;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double t1
//                const double t2
//                const int p2
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel12(const
                                                                     double t1, const double t2, const int p2,
                                                                     double Tinv[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[p2] = -(1.0305949982581226 * t2 + -0.22205377950113064 * t1) /
                   1.1512965678044422;
        Tinv[p2 + 1] = t2;
        Tinv[p2 + 2] = t1;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double t1
//                const double t2
//                const int p3
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel13(const
                                                                     double t1, const double t2, const int p3,
                                                                     double Tinv[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[p3] = -(1.0305949982581226 * t2 + -0.22205377950113064 * t1) /
                   1.1512965678044422;
        Tinv[p3 + 1] = t2;
        Tinv[p3 + 2] = t1;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float lt_y[28]
//                float fv[84]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel14(const float
                                                                     lt_y[28], float fv[84]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 28) {
        fv[k] = static_cast<float>(k) + 3.0F;
        fv[k + 28] = lt_y[k];
        fv[k + 56] = 1.0F;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Tinv[9]
//                const float fv[84]
//                float U[84]
// Return Type  : void
//
static __global__ __launch_bounds__(96, 1) void detect_lane_kernel15(const
                                                                     double Tinv[9], const float fv[84], float U[84]) {
    unsigned long threadId;
    float f;
    int ibcol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    ibcol = static_cast<int>(threadId % 3UL);
    k = static_cast<int>((threadId - static_cast<unsigned long>(ibcol)) / 3UL);
    if ((static_cast<int>(k < 28)) && (static_cast<int>(ibcol < 3))) {
        f = 0.0F;
        for (int jtilecol = 0; jtilecol < 3; jtilecol++) {
            f += fv[k + 28 * jtilecol] * static_cast<float>(Tinv[jtilecol + 3 * ibcol]);
        }

        U[k + 28 * ibcol] = f;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float U[84]
//                float b[56]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void detect_lane_kernel16(const float
                                                                     U[84], float b[56]) {
    unsigned long threadId;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    k = static_cast<int>(threadId % 28UL);
    jtilecol = static_cast<int>((threadId - static_cast<unsigned long>(k)) / 28UL);
    if ((static_cast<int>(jtilecol < 2)) && (static_cast<int>(k < 28))) {
        b[jtilecol * 28 + k] = U[k + 56];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float b[56]
//                float U[84]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void detect_lane_kernel17(const float
                                                                     b[56], float U[84]) {
    unsigned long threadId;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    jtilecol = static_cast<int>(threadId % 28UL);
    k = static_cast<int>((threadId - static_cast<unsigned long>(jtilecol)) / 28UL);
    if ((static_cast<int>(k < 2)) && (static_cast<int>(jtilecol < 28))) {
        U[jtilecol + 28 * k] /= b[jtilecol + 28 * k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float U[84]
//                float ltPts[56]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void detect_lane_kernel18(const float
                                                                     U[84], float ltPts[56]) {
    unsigned long threadId;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    jtilecol = static_cast<int>(threadId % 28UL);
    k = static_cast<int>((threadId - static_cast<unsigned long>(jtilecol)) / 28UL);
    if ((static_cast<int>(k < 2)) && (static_cast<int>(jtilecol < 28))) {
        ltPts[jtilecol + 28 * k] = U[jtilecol + 28 * k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Tinv[9]
//                double b_x[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel19(const
                                                                     double Tinv[9], double b_x[9]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 9) {
        b_x[k] = Tinv[k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double laneCoeffMeans[6]
//                const double laneCoeffStds[6]
//                float lanecoeffsNetworkOutput[6]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel2(const double
                                                                    laneCoeffMeans[6], const double laneCoeffStds[6],
                                                                    float
                                                                    lanecoeffsNetworkOutput[6]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 6) {
        //  Recover original coeffs by reversing the normalization steps
        lanecoeffsNetworkOutput[k] = lanecoeffsNetworkOutput[k] * static_cast<float>
        (laneCoeffStds[k]) + static_cast<float>(laneCoeffMeans[k]);
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double b_x[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel20(double b_x
[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        b_x[1] /= 1.1512965678044422;
        b_x[2] = -1.718788847108661E-19;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double b_x[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel21(double b_x
[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        b_x[4] -= b_x[1] * 1.0305949982581226;
        b_x[5] = -0.0032316839464807288 - b_x[2] * 1.0305949982581226;
        b_x[7] -= b_x[1] * -0.22205377950113064;
        b_x[8] = 1.9916790026632809E-35 - b_x[2] * -0.22205377950113064;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double t1
//                const double t2
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel22(const
                                                                     double t1, const double t2, double Tinv[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[6] = ((1.0 - 1.0305949982581226 * t2) - -0.22205377950113064 * t1) /
                  1.1512965678044422;
        Tinv[7] = t2;
        Tinv[8] = t1;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double t1
//                const double t2
//                const int p2
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel23(const
                                                                     double t1, const double t2, const int p2,
                                                                     double Tinv[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[p2] = -(1.0305949982581226 * t2 + -0.22205377950113064 * t1) /
                   1.1512965678044422;
        Tinv[p2 + 1] = t2;
        Tinv[p2 + 2] = t1;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double t1
//                const double t2
//                const int p3
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel24(const
                                                                     double t1, const double t2, const int p3,
                                                                     double Tinv[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[p3] = -(1.0305949982581226 * t2 + -0.22205377950113064 * t1) /
                   1.1512965678044422;
        Tinv[p3 + 1] = t2;
        Tinv[p3 + 2] = t1;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float rt_y[28]
//                float fv[84]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel25(const float
                                                                     rt_y[28], float fv[84]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 28) {
        fv[k] = static_cast<float>(k) + 3.0F;
        fv[k + 28] = rt_y[k];
        fv[k + 56] = 1.0F;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Tinv[9]
//                const float fv[84]
//                float U[84]
// Return Type  : void
//
static __global__ __launch_bounds__(96, 1) void detect_lane_kernel26(const
                                                                     double Tinv[9], const float fv[84], float U[84]) {
    unsigned long threadId;
    float f;
    int ibcol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    ibcol = static_cast<int>(threadId % 3UL);
    k = static_cast<int>((threadId - static_cast<unsigned long>(ibcol)) / 3UL);
    if ((static_cast<int>(k < 28)) && (static_cast<int>(ibcol < 3))) {
        f = 0.0F;
        for (int jtilecol = 0; jtilecol < 3; jtilecol++) {
            f += fv[k + 28 * jtilecol] * static_cast<float>(Tinv[jtilecol + 3 * ibcol]);
        }

        U[k + 28 * ibcol] = f;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float U[84]
//                float b[56]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void detect_lane_kernel27(const float
                                                                     U[84], float b[56]) {
    unsigned long threadId;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    k = static_cast<int>(threadId % 28UL);
    jtilecol = static_cast<int>((threadId - static_cast<unsigned long>(k)) / 28UL);
    if ((static_cast<int>(jtilecol < 2)) && (static_cast<int>(k < 28))) {
        b[jtilecol * 28 + k] = U[k + 56];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float b[56]
//                float U[84]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void detect_lane_kernel28(const float
                                                                     b[56], float U[84]) {
    unsigned long threadId;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    jtilecol = static_cast<int>(threadId % 28UL);
    k = static_cast<int>((threadId - static_cast<unsigned long>(jtilecol)) / 28UL);
    if ((static_cast<int>(k < 2)) && (static_cast<int>(jtilecol < 28))) {
        U[jtilecol + 28 * k] /= b[jtilecol + 28 * k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float U[84]
//                float rtPts[56]
// Return Type  : void
//
static __global__ __launch_bounds__(64, 1) void detect_lane_kernel29(const float
                                                                     U[84], float rtPts[56]) {
    unsigned long threadId;
    int jtilecol;
    int k;
    threadId = mwGetGlobalThreadIndex();
    jtilecol = static_cast<int>(threadId % 28UL);
    k = static_cast<int>((threadId - static_cast<unsigned long>(jtilecol)) / 28UL);
    if ((static_cast<int>(k < 2)) && (static_cast<int>(jtilecol < 28))) {
        rtPts[jtilecol + 28 * k] = U[jtilecol + 28 * k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float lanecoeffsNetworkOutput[6]
//                float rt_y[28]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel3(const float
                                                                    lanecoeffsNetworkOutput[6], float rt_y[28]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 28) {
        rt_y[k] = lanecoeffsNetworkOutput[3];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float lanecoeffsNetworkOutput
//                float rt_y[28]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel4(const float
                                                                    lanecoeffsNetworkOutput, float rt_y[28]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 28) {
        rt_y[k] = (static_cast<float>(k) + 3.0F) * rt_y[k] + lanecoeffsNetworkOutput;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float lanecoeffsNetworkOutput[6]
//                float lt_y[28]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel5(const float
                                                                    lanecoeffsNetworkOutput[6], float lt_y[28]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 28) {
        lt_y[k] = lanecoeffsNetworkOutput[0];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const float lanecoeffsNetworkOutput
//                float lt_y[28]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel6(const float
                                                                    lanecoeffsNetworkOutput, float lt_y[28]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 28) {
        lt_y[k] = (static_cast<float>(k) + 3.0F) * lt_y[k] + lanecoeffsNetworkOutput;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double Tinv[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel7(double Tinv
[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        Tinv[2] = 1.1512965678044422;
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                const double Tinv[9]
//                double b_x[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel8(const double
                                                                    Tinv[9], double b_x[9]) {
    int k;
    k = static_cast<int>(mwGetGlobalThreadIndex());
    if (k < 9) {
        //  map vehicle to image coordinates
        b_x[k] = Tinv[k];
    }
}

//
// Arguments    : dim3 blockArg
//                dim3 gridArg
//                double b_x[9]
// Return Type  : void
//
static __global__ __launch_bounds__(32, 1) void detect_lane_kernel9(double b_x[9]) {
    int tmpIdx;
    tmpIdx = static_cast<int>(mwGetGlobalThreadIndex());
    if (tmpIdx < 1) {
        b_x[1] /= 1.1512965678044422;
        b_x[2] = -1.718788847108661E-19;
    }
}

//
// From the networks output, compute left and right lane points in the
//  image coordinates. The camera coordinates are described by the caltech
//  mono camera model.
// Arguments    : const float frame[154587]
//                const double laneCoeffMeans[6]
//                const double laneCoeffStds[6]
//                boolean_T *laneFound
//                float ltPts[56]
//                float rtPts[56]
// Return Type  : void
//
void detect_lane(const float frame[154587], const double laneCoeffMeans[6],
                 const double laneCoeffStds[6], boolean_T *laneFound, float
                 ltPts[56], float rtPts[56]) {
    static float b[154587];
    double Tinv[9];
    double b_x[9];
    double (*b_gpu_Tinv)[9];
    double (*gpu_Tinv)[9];
    double (*gpu_x)[9];
    double (*gpu_laneCoeffMeans)[6];
    double (*gpu_laneCoeffStds)[6];
    float (*gpu_b)[154587];
    float (*gpu_frame)[154587];
    float (*gpu_U)[84];
    float (*gpu_fv)[84];
    float (*b_gpu_b)[56];
    float (*gpu_ltPts)[56];
    float (*gpu_rtPts)[56];
    float (*gpu_lt_y)[28];
    float (*gpu_rt_y)[28];
    float lanecoeffsNetworkOutput[6];
    float (*gpu_lanecoeffsNetworkOutput)[6];
    boolean_T rtPts_dirtyOnGpu;
    if (!isInitialized_detect_lane) {
        detect_lane_initialize();
    }

    hipMalloc(&gpu_rtPts, 224UL);
    hipMalloc(&gpu_ltPts, 224UL);
    hipMalloc(&b_gpu_b, 224UL);
    hipMalloc(&gpu_U, 336UL);
    hipMalloc(&gpu_fv, 336UL);
    hipMalloc(&b_gpu_Tinv, 72UL);
    hipMalloc(&gpu_x, 72UL);
    hipMalloc(&gpu_Tinv, 72UL);
    hipMalloc(&gpu_lt_y, 112UL);
    hipMalloc(&gpu_rt_y, 112UL);
    hipMalloc(&gpu_lanecoeffsNetworkOutput, 24UL);
    hipMalloc(&gpu_laneCoeffStds, 48UL);
    hipMalloc(&gpu_laneCoeffMeans, 48UL);
    hipMalloc(&gpu_b, 618348UL);
    hipMalloc(&gpu_frame, 618348UL);
    rtPts_dirtyOnGpu = false;

    //  A persistent object mynet is used to load the series network object.
    //  At the first call to this function, the persistent object is constructed and
    //  setup. When the function is called subsequent times, the same object is reused
    //  to call predict on inputs, thus avoiding reconstructing and reloading the
    //  network object.
    if (!lanenet_not_empty) {
        coder::DeepLearningNetwork_setup(&lanenet);
        lanenet_not_empty = true;
    }

    hipMemcpy(gpu_frame, (void *) &frame[0], 618348UL, hipMemcpyHostToDevice);
    detect_lane_kernel1<<<dim3(302U, 1U, 1U), dim3(512U, 1U, 1U)>>>(*gpu_frame,
                                                                    *gpu_b);
    hipMemcpy(&b[0], gpu_b, 618348UL, hipMemcpyDeviceToHost);
    coder::DeepLearningNetwork_predict(&lanenet, b, lanecoeffsNetworkOutput);

    //  Recover original coeffs by reversing the normalization steps
    hipMemcpy(gpu_laneCoeffMeans, (void *) &laneCoeffMeans[0], 48UL,
               hipMemcpyHostToDevice);
    hipMemcpy(gpu_laneCoeffStds, (void *) &laneCoeffStds[0], 48UL,
               hipMemcpyHostToDevice);
    hipMemcpy(gpu_lanecoeffsNetworkOutput, &lanecoeffsNetworkOutput[0], 24UL,
               hipMemcpyHostToDevice);
    detect_lane_kernel2<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
            (*gpu_laneCoeffMeans, *gpu_laneCoeffStds, *gpu_lanecoeffsNetworkOutput);

    // c should be more than 0.5 for it to be a right lane
    // meters, ahead of the sensor
    hipMemcpy(&lanecoeffsNetworkOutput[0], gpu_lanecoeffsNetworkOutput, 24UL,
               hipMemcpyDeviceToHost);
    if ((std::abs(lanecoeffsNetworkOutput[5]) > 0.5F) && (std::abs
                                                                  (lanecoeffsNetworkOutput[2]) > 0.5F)) {
        double t1;
        int p2;
        int p3;
        detect_lane_kernel3<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
                (*gpu_lanecoeffsNetworkOutput, *gpu_rt_y);
        for (p2 = 0; p2 < 2; p2++) {
            detect_lane_kernel4<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
                    (lanecoeffsNetworkOutput[p2 + 4], *gpu_rt_y);
        }

        detect_lane_kernel5<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
                (*gpu_lanecoeffsNetworkOutput, *gpu_lt_y);
        for (p2 = 0; p2 < 2; p2++) {
            detect_lane_kernel6<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>
                    (lanecoeffsNetworkOutput[p2 + 1], *gpu_lt_y);
        }

        //  Visualize lane boundaries of the ego vehicle
        //  Compute extrinsics based on camera setup
        //  pitch of the camera in degrees
        //  Construct a camera matrix
        //  Turn camMatrix into 2-D homography
        //  drop Z
        Tinv[3] = -0.0032316839464807288;
        Tinv[4] = -1.2852132429203174E-19;
        Tinv[5] = 1.0305949982581226;
        Tinv[6] = 1.9916790026632809E-35;
        Tinv[7] = 0.0012931719938928032;
        Tinv[8] = -0.22205377950113064;
        Tinv[0] = -1.9788357004567556E-19;
        Tinv[1] = -0.00070281981464454381;
        hipMemcpy(gpu_Tinv, &Tinv[0], 72UL, hipMemcpyHostToDevice);
        detect_lane_kernel7<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_Tinv);

        //  map vehicle to image coordinates
        detect_lane_kernel8<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_Tinv,
                                                                     *gpu_x);
        p2 = 3;
        p3 = 0;
        detect_lane_kernel9<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x);
        detect_lane_kernel10<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x);
        hipMemcpy(&b_x[0], gpu_x, 72UL, hipMemcpyDeviceToHost);
        if (std::abs(b_x[5]) > std::abs(b_x[4])) {
            p2 = 0;
            p3 = 3;
            t1 = b_x[1];
            b_x[1] = b_x[2];
            b_x[2] = t1;
            t1 = b_x[4];
            b_x[4] = b_x[5];
            b_x[5] = t1;
            t1 = b_x[7];
            b_x[7] = b_x[8];
            b_x[8] = t1;
        }

        b_x[5] /= b_x[4];
        b_x[8] -= b_x[5] * b_x[7];
        t1 = (b_x[5] * b_x[1] - b_x[2]) / b_x[8];
        detect_lane_kernel11<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(t1, -(b_x[1] +
                                                                            b_x[7] * t1) / b_x[4], *b_gpu_Tinv);
        t1 = -b_x[5] / b_x[8];
        detect_lane_kernel12<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(t1, (1.0 -
                                                                           b_x[7] * t1) / b_x[4], p2, *b_gpu_Tinv);
        t1 = 1.0 / b_x[8];
        detect_lane_kernel13<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(t1, -b_x[7] *
                                                                          t1 / b_x[4], p3, *b_gpu_Tinv);
        detect_lane_kernel14<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_lt_y,
                                                                      *gpu_fv);
        detect_lane_kernel15<<<dim3(1U, 1U, 1U), dim3(96U, 1U, 1U)>>>(*b_gpu_Tinv,
                                                                      *gpu_fv, *gpu_U);
        detect_lane_kernel16<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_U,
                                                                      *b_gpu_b);
        detect_lane_kernel17<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*b_gpu_b,
                                                                      *gpu_U);
        detect_lane_kernel18<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_U,
                                                                      *gpu_ltPts);
        hipMemcpy(gpu_x, &b_x[0], 72UL, hipMemcpyHostToDevice);
        detect_lane_kernel19<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_Tinv,
                                                                      *gpu_x);
        p2 = 3;
        p3 = 0;
        detect_lane_kernel20<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x);
        detect_lane_kernel21<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_x);
        hipMemcpy(&b_x[0], gpu_x, 72UL, hipMemcpyDeviceToHost);
        if (std::abs(b_x[5]) > std::abs(b_x[4])) {
            p2 = 0;
            p3 = 3;
            t1 = b_x[1];
            b_x[1] = b_x[2];
            b_x[2] = t1;
            t1 = b_x[4];
            b_x[4] = b_x[5];
            b_x[5] = t1;
            t1 = b_x[7];
            b_x[7] = b_x[8];
            b_x[8] = t1;
        }

        b_x[5] /= b_x[4];
        b_x[8] -= b_x[5] * b_x[7];
        t1 = (b_x[5] * b_x[1] - b_x[2]) / b_x[8];
        detect_lane_kernel22<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(t1, -(b_x[1] +
                                                                            b_x[7] * t1) / b_x[4], *gpu_Tinv);
        t1 = -b_x[5] / b_x[8];
        detect_lane_kernel23<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(t1, (1.0 -
                                                                           b_x[7] * t1) / b_x[4], p2, *gpu_Tinv);
        t1 = 1.0 / b_x[8];
        detect_lane_kernel24<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(t1, -b_x[7] *
                                                                          t1 / b_x[4], p3, *gpu_Tinv);
        detect_lane_kernel25<<<dim3(1U, 1U, 1U), dim3(32U, 1U, 1U)>>>(*gpu_rt_y,
                                                                      *gpu_fv);
        detect_lane_kernel26<<<dim3(1U, 1U, 1U), dim3(96U, 1U, 1U)>>>(*gpu_Tinv,
                                                                      *gpu_fv, *gpu_U);
        detect_lane_kernel27<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_U,
                                                                      *b_gpu_b);
        detect_lane_kernel28<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*b_gpu_b,
                                                                      *gpu_U);
        detect_lane_kernel29<<<dim3(1U, 1U, 1U), dim3(64U, 1U, 1U)>>>(*gpu_U,
                                                                      *gpu_rtPts);
        rtPts_dirtyOnGpu = true;
        *laneFound = true;
        hipMemcpy(&ltPts[0], gpu_ltPts, 224UL, hipMemcpyDeviceToHost);
    } else {
        *laneFound = false;
    }

    if (rtPts_dirtyOnGpu) {
        hipMemcpy(&rtPts[0], gpu_rtPts, 224UL, hipMemcpyDeviceToHost);
    }

    hipFree(*gpu_frame);
    hipFree(*gpu_b);
    hipFree(*gpu_laneCoeffMeans);
    hipFree(*gpu_laneCoeffStds);
    hipFree(*gpu_lanecoeffsNetworkOutput);
    hipFree(*gpu_rt_y);
    hipFree(*gpu_lt_y);
    hipFree(*gpu_Tinv);
    hipFree(*gpu_x);
    hipFree(*b_gpu_Tinv);
    hipFree(*gpu_fv);
    hipFree(*gpu_U);
    hipFree(*b_gpu_b);
    hipFree(*gpu_ltPts);
    hipFree(*gpu_rtPts);
}

//
// From the networks output, compute left and right lane points in the
//  image coordinates. The camera coordinates are described by the caltech
//  mono camera model.
// Arguments    : void
// Return Type  : void
//
void detect_lane_init() {
    lanenet_not_empty = false;
}

//
// File trailer for detect_lane.cu
//
// [EOF]
//
