//
// File: DeepLearningNetwork.cu
//
// GPU Coder version                    : 2.0
// CUDA/C/C++ source code generated on  : 15-Dec-2020 12:44:50
//

// Include Files
#include "DeepLearningNetwork.h"
#include "detect_lane_internal_types.h"
#include "MWElementwiseAffineLayer.hpp"
#include "MWFusedConvReLULayer.hpp"
#include "MWTargetNetworkImpl.hpp"
#include "cnn_api.hpp"
#include <cstdio>

const char *errorString =
        "Abnormal termination due to: %s.\nError in %s (line %d).";

// Function Declarations
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line);

// Function Definitions
//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::allocate() {
    this->targetImpl->allocate(290400, 2);
    for (int idx = 0; idx < 18; idx++) {
        this->layers[idx]->allocate();
    }

    (static_cast<MWTensor<float> *>(this->inputTensors[0]))->setData(this->layers
                                                                     [0]->getLayerOutput(0));
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::cleanup() {
    this->deallocate();
    for (int idx = 0; idx < 18; idx++) {
        this->layers[idx]->cleanup();
    }

    if (this->targetImpl) {
        this->targetImpl->cleanup();
    }
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::deallocate() {
    this->targetImpl->deallocate();
    for (int idx = 0; idx < 18; idx++) {
        this->layers[idx]->deallocate();
    }
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::postsetup() {
    this->targetImpl->postSetup(this->layers, this->numLayers);
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::setSize() {
    for (int idx = 0; idx < 18; idx++) {
        this->layers[idx]->propagateSize();
    }

    this->allocate();
    this->postsetup();
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::setup() {
    this->targetImpl->preSetup();
    this->targetImpl->setAutoTune(true);
    (static_cast<MWInputLayer *>(this->layers[0]))->createInputLayer
            (this->targetImpl, this->inputTensors[0], 227, 227, 3, 0, "", 0);
    (static_cast<MWElementwiseAffineLayer *>(this->layers[1]))
            ->createElementwiseAffineLayer(this->targetImpl, this->layers[0]
                                                   ->getOutputTensor(0), 227, 227, 3, 227, 227, 3, false, 1, 1,
                                           "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_data_scale.bin",
                                           "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_data_offset.bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[2]))
            ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[1]
                                               ->getOutputTensor(0), 11, 11, 3, 96, 4, 4, 0, 0, 0, 0, 1, 1, 1,
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv1_w.bin",
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv1_b.bin", 1);
    (static_cast<MWNormLayer *>(this->layers[3]))->createNormLayer
            (this->targetImpl, this->layers[2]->getOutputTensor(0), 5, 0.0001, 0.75, 1.0,
             0);
    (static_cast<MWMaxPoolingLayer *>(this->layers[4]))->createMaxPoolingLayer
            (this->targetImpl, this->layers[3]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0,
             0, 0, 1, 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[5]))
            ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[4]
                                               ->getOutputTensor(0), 5, 5, 48, 128, 1, 1, 2, 2, 2, 2, 1, 1, 2,
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv2_w.bin",
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv2_b.bin", 0);
    (static_cast<MWNormLayer *>(this->layers[6]))->createNormLayer
            (this->targetImpl, this->layers[5]->getOutputTensor(0), 5, 0.0001, 0.75, 1.0,
             1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[7]))->createMaxPoolingLayer
            (this->targetImpl, this->layers[6]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0,
             0, 0, 1, 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[8]))
            ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[7]
                                               ->getOutputTensor(0), 3, 3, 256, 384, 1, 1, 1, 1, 1, 1, 1, 1, 1,
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv3_w.bin",
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv3_b.bin", 1);
    (static_cast<MWFusedConvReLULayer *>(this->layers[9]))
            ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[8]
                                               ->getOutputTensor(0), 3, 3, 192, 192, 1, 1, 1, 1, 1, 1, 1, 1, 2,
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv4_w.bin",
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv4_b.bin", 0);
    (static_cast<MWFusedConvReLULayer *>(this->layers[10]))
            ->createFusedConvReLULayer(this->targetImpl, 1, this->layers[9]
                                               ->getOutputTensor(0), 3, 3, 192, 128, 1, 1, 1, 1, 1, 1, 1, 1, 2,
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv5_w.bin",
                                       "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_conv5_b.bin", 1);
    (static_cast<MWMaxPoolingLayer *>(this->layers[11]))->createMaxPoolingLayer
            (this->targetImpl, this->layers[10]->getOutputTensor(0), 3, 3, 2, 2, 0, 0, 0,
             0, 0, 1, 0);
    (static_cast<MWFCLayer *>(this->layers[12]))->createFCLayer(this->targetImpl,
                                                                this->layers[11]->getOutputTensor(0), 9216, 4096,
                                                                "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_fc6_w.bin",
                                                                "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_fc6_b.bin",
                                                                1);
    (static_cast<MWReLULayer *>(this->layers[13]))->createReLULayer
            (this->targetImpl, this->layers[12]->getOutputTensor(0), 1);
    (static_cast<MWFCLayer *>(this->layers[14]))->createFCLayer(this->targetImpl,
                                                                this->layers[13]->getOutputTensor(0), 4096, 16,
                                                                "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_fcLane1_w.bin",
                                                                "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_fcLane1_b.bin",
                                                                0);
    (static_cast<MWReLULayer *>(this->layers[15]))->createReLULayer
            (this->targetImpl, this->layers[14]->getOutputTensor(0), 0);
    (static_cast<MWFCLayer *>(this->layers[16]))->createFCLayer(this->targetImpl,
                                                                this->layers[15]->getOutputTensor(0), 16, 6,
                                                                "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_fcLane2_w.bin",
                                                                "/home/dinhnambkhn/CLionProjects/GPUDeep_lanenet/detect_lane/cnn_lanenet0_0_fcLane2_b.bin",
                                                                1);
    (static_cast<MWOutputLayer *>(this->layers[17]))->createOutputLayer
            (this->targetImpl, this->layers[16]->getOutputTensor(0), 1);
    this->outputTensors[0] = this->layers[17]->getOutputTensor(0);
    this->setSize();
}

//
// Arguments    : hipError_t errCode
//                const char *file
//                unsigned int line
// Return Type  : void
//
static void checkCleanupCudaError(hipError_t errCode, const char *file,
                                  unsigned int line) {
    if ((errCode != hipSuccess) && (errCode != hipErrorDeinitialized)) {
        printf(errorString, hipGetErrorString(errCode), file, line);
    }
}

//
// Arguments    : void
// Return Type  : int
//
int lanenet0_0::getBatchSize() {
    return this->inputTensors[0]->getBatchSize();
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *lanenet0_0::getInputDataPointer(int b_index) {
    return (static_cast<MWTensor<float> *>(this->inputTensors[b_index]))->getData();
}

//
// Arguments    : void
// Return Type  : float *
//
float *lanenet0_0::getInputDataPointer() {
    return (static_cast<MWTensor<float> *>(this->inputTensors[0]))->getData();
}

//
// Arguments    : int layerIndex
//                int portIndex
// Return Type  : float *
//
float *lanenet0_0::getLayerOutput(int layerIndex, int portIndex) {
    return this->layers[layerIndex]->getLayerOutput(portIndex);
}

//
// Arguments    : int b_index
// Return Type  : float *
//
float *lanenet0_0::getOutputDataPointer(int b_index) {
    return (static_cast<MWTensor<float> *>(this->outputTensors[b_index]))->getData
            ();
}

//
// Arguments    : void
// Return Type  : float *
//
float *lanenet0_0::getOutputDataPointer() {
    return (static_cast<MWTensor<float> *>(this->outputTensors[0]))->getData();
}

//
// Arguments    : void
// Return Type  : void
//
lanenet0_0::lanenet0_0() {
    this->numLayers = 18;
    this->targetImpl = 0;
    this->layers[0] = new MWInputLayer;
    this->layers[0]->setName("data");
    this->layers[1] = new MWElementwiseAffineLayer;
    this->layers[1]->setName("data_normalization");
    this->layers[1]->setInPlaceIndex(0, 0);
    this->layers[2] = new MWFusedConvReLULayer;
    this->layers[2]->setName("conv1_relu1");
    this->layers[3] = new MWNormLayer;
    this->layers[3]->setName("norm1");
    this->layers[4] = new MWMaxPoolingLayer;
    this->layers[4]->setName("pool1");
    this->layers[5] = new MWFusedConvReLULayer;
    this->layers[5]->setName("conv2_relu2");
    this->layers[6] = new MWNormLayer;
    this->layers[6]->setName("norm2");
    this->layers[7] = new MWMaxPoolingLayer;
    this->layers[7]->setName("pool2");
    this->layers[8] = new MWFusedConvReLULayer;
    this->layers[8]->setName("conv3_relu3");
    this->layers[9] = new MWFusedConvReLULayer;
    this->layers[9]->setName("conv4_relu4");
    this->layers[10] = new MWFusedConvReLULayer;
    this->layers[10]->setName("conv5_relu5");
    this->layers[11] = new MWMaxPoolingLayer;
    this->layers[11]->setName("pool5");
    this->layers[12] = new MWFCLayer;
    this->layers[12]->setName("fc6");
    this->layers[13] = new MWReLULayer;
    this->layers[13]->setName("relu6");
    this->layers[13]->setInPlaceIndex(0, 0);
    this->layers[14] = new MWFCLayer;
    this->layers[14]->setName("fcLane1");
    this->layers[15] = new MWReLULayer;
    this->layers[15]->setName("fcLane1Relu");
    this->layers[15]->setInPlaceIndex(0, 0);
    this->layers[16] = new MWFCLayer;
    this->layers[16]->setName("fcLane2");
    this->layers[17] = new MWOutputLayer;
    this->layers[17]->setName("output");
    this->layers[17]->setInPlaceIndex(0, 0);
    this->targetImpl = new MWTargetNetworkImpl;
    this->inputTensors[0] = new MWTensor<float>;
    this->inputTensors[0]->setHeight(227);
    this->inputTensors[0]->setWidth(227);
    this->inputTensors[0]->setChannels(3);
    this->inputTensors[0]->setBatchSize(1);
    this->inputTensors[0]->setSequenceLength(1);
}

//
// Arguments    : void
// Return Type  : void
//
lanenet0_0::~lanenet0_0() {
    this->cleanup();
    checkCleanupCudaError(hipGetLastError(), __FILE__, __LINE__);
    for (int idx = 0; idx < 18; idx++) {
        delete this->layers[idx];
    }

    if (this->targetImpl) {
        delete this->targetImpl;
    }

    delete this->inputTensors[0];
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::predict() {
    for (int idx = 0; idx < 18; idx++) {
        this->layers[idx]->predict();
    }
}

//
// Arguments    : void
// Return Type  : void
//
void lanenet0_0::resetState() {
}

//
// Arguments    : lanenet0_0 *obj
// Return Type  : void
//
namespace coder {
    void DeepLearningNetwork_setup(lanenet0_0 *obj) {
        obj->setup();
    }
}

//
// File trailer for DeepLearningNetwork.cu
//
// [EOF]
//
